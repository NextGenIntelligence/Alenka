#include "hip/hip_runtime.h"
// PFOR and PFOR-DELTA Compression and decompression routines

#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <exception>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/extrema.h>
#include "sorts.cu"

using namespace std;

struct bool_to_int
{
    __host__ __device__
    unsigned int operator()(const bool x)
    {
        return (unsigned int)x;
    }
};

struct ui_to_ll
{
    __host__ __device__
    long long int operator()(const unsigned int x)
    {
        return (long long int)x;
    }
};

template<typename T>
struct nz
{
    __host__ __device__
    bool operator()(const T x)
    {
        return (x != 0);
    }
};


struct compress_functor_int
{

    const int_type * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_int(const int_type * _source, unsigned long long int  * _dest,
                         const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val = source[i] - start_val[0];;
        unsigned int shifted = vals[2] - vals[0] - (i%vals[1])*vals[0];
        dest[i] = val << shifted;
    }
};

struct compress_functor_float
{

    const long long int * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_float(const long long int * _source, unsigned long long int  * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val;

        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];

        val = source[i] - start_val[0];
        unsigned int z = i%fit_count;

        unsigned int shifted = int_sz - bits - z*bits;
        dest[i] = val << shifted;
    }
};



struct decompress_functor_int
{

    const unsigned long long int * source;
    int_type * dest;
    const long long int * start_val;
    const unsigned int * vals;


    decompress_functor_int(const unsigned long long int * _source, int_type * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {


        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = int_sz - bits - src_loc*bits;
        unsigned long long int tmp = source[src_idx]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);

        dest[i] = tmp + start_val[0];

    }
};


struct decompress_functor_float
{

    const unsigned long long int * source;
    long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    decompress_functor_float(const unsigned long long int * _source, long long int * _dest,
                             const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {


        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = int_sz - bits - src_loc*bits;
        unsigned long long int tmp = source[src_idx]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);

        dest[i] = tmp + start_val[0];

    }
};

void resize_compressed(void*& host, unsigned long long int host_sz, unsigned long long int sz, bool cuda_delete) // sz and host_sz in bytes
{
    char* n;
	
	if (cuda_delete)	
        hipHostMalloc(&n, host_sz + sz);
	else
        n = new char[host_sz + sz];
		

    //if(errVal != hipSuccess) {
    //    cout << "ALLOC ERROR " << hipGetErrorString(errVal) << endl;
    //    exit(0);
    //};
	
	if(host_sz) {
	    memcpy(n,host,host_sz);
	    if (cuda_delete)
            hipHostFree(host);
	    else
            delete [] host;	
	};		

    host = n;
}



long long int pfor_dict_decompress(void* compressed, char** h_columns, char** d_columns, unsigned int* mRecCount, FILE* f, bool mode, unsigned int mColumnCount, unsigned int offset)
{

    unsigned int bits, cnt, fit_count, orig_recCount, grp_count;
    long long int  orig_lower_val;
    unsigned int bit_count = 64;
    unsigned int comp_type;
    long long int start_val;
	

	if (f) {
        fread((char *)&grp_count, 4, 1, f);
        fread((char *)&cnt, 4, 1, f);
        fread((char *)&orig_recCount, 4, 1, f);
        fread((char *)&bits, 4, 1, f);
        fread((char *)&orig_lower_val, 8, 1, f);
        fread((char *)&fit_count, 4, 1, f);
        fread((char *)&start_val, 8, 1, f);
        fread((char *)&comp_type, 4, 1, f);
	}
    else {
	    cnt = ((unsigned int*)compressed)[0];
		grp_count = ((unsigned int*)((char*)compressed + 8*cnt + 12))[0];			  
        orig_recCount = ((unsigned int*)((char*)compressed + 8*cnt +8))[0];			  
        bits = ((unsigned int*)((char*)compressed + 8*cnt + mColumnCount*grp_count + 28))[0];			  
        orig_lower_val = ((long long int*)((char*)compressed + 8*cnt + mColumnCount*grp_count + 32))[0];			  
        fit_count = ((unsigned int*)((char*)compressed + 8*cnt + mColumnCount*grp_count + 40))[0];			  
        start_val = ((long long int*)((char*)compressed + 8*cnt + mColumnCount*grp_count + 44))[0];			  
        comp_type  = ((unsigned int*)((char*)compressed + 8*cnt + mColumnCount*grp_count + 52))[0];			  
    };	
	*mRecCount = orig_recCount;
   

    //cout << "DICT Decomp Header " << cnt << " " << grp_count << " " << orig_recCount << " " << bits << " " << orig_lower_val << " " << fit_count << " " << start_val << " " << comp_type  << endl;


    thrust::device_ptr<unsigned long long int> decomp = thrust::device_malloc<unsigned long long int>(cnt);
    unsigned long long int* raw_decomp = thrust::raw_pointer_cast(decomp);
	if (f)
        hipMemcpy( (void*)raw_decomp, (void*)compressed, cnt*8, hipMemcpyHostToDevice);
	else 
       	hipMemcpy( (void*)raw_decomp, (void*)((unsigned int*)compressed + 1), cnt*8, hipMemcpyHostToDevice);

    void* d_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);

    void* s_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);



    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;

    thrust::device_ptr<unsigned long long int> dest = thrust::device_malloc<unsigned long long int>(orig_recCount);


    thrust::counting_iterator<unsigned int, thrust::device_system_tag> begin(0);
    decompress_functor_int ff1(raw_decomp,(int_type*)thrust::raw_pointer_cast(dest), (long long int*)s_v, (unsigned int*)d_v);
    thrust::for_each(begin, begin + orig_recCount, ff1);

    hipFree(d_v);
    hipFree(s_v);
    thrust::device_free(decomp);
	
	
    if(mode == 0) {                   // keep results in gpu

        thrust::device_ptr<char> dict = thrust::device_malloc<char>(grp_count);

        for(unsigned int i = 0; i < mColumnCount; i++) {
            thrust::device_ptr<char> d_col((char*)d_columns[i] + offset);
			if(f)
                hipMemcpy( (void*)thrust::raw_pointer_cast(dict), (void*)(h_columns[i] + offset) , grp_count, hipMemcpyHostToDevice);
			else 
			    hipMemcpy( (void*)thrust::raw_pointer_cast(dict), (void*)((char*)compressed + 8*cnt + 16 + i*grp_count) , grp_count, hipMemcpyHostToDevice);                			
            thrust::gather(dest, dest+orig_recCount,dict, d_col);
        }
        thrust::device_free(dict);
    }
    else {
        thrust::device_ptr<char> dict = thrust::device_malloc<char>(grp_count);
        thrust::device_ptr<char> d_col = thrust::device_malloc<char>(orig_recCount);

        for(unsigned int i = 0; i < mColumnCount; i++) {
            hipMemcpy( (void*)thrust::raw_pointer_cast(dict), (void*)(h_columns[i] + offset), grp_count, hipMemcpyHostToDevice);
            thrust::gather(dest, dest+orig_recCount,dict, d_col);
            hipMemcpy((void*)(h_columns[i] +offset), thrust::raw_pointer_cast(d_col), orig_recCount, hipMemcpyDeviceToHost);
        }
        thrust::device_free(dict);
        thrust::device_free(d_col);

    };
    thrust::device_free(dest);
	
    return 1;
}






long long int pfor_decompress(void* destination, void* host, unsigned int* mRecCount, bool tp, FILE* f)
{

    unsigned int bits, cnt, fit_count, orig_recCount;
    long long int  orig_lower_val;
    unsigned int bit_count = 64;
    unsigned int comp_type;
    long long int start_val;

	if(f) {
        fread((char *)&cnt, 4, 1, f);
        fread((char *)&cnt, 4, 1, f);
        fread((char *)&orig_recCount, 4, 1, f);
        fread((char *)&bits, 4, 1, f);
        fread((char *)&orig_lower_val, 8, 1, f);
        fread((char *)&fit_count, 4, 1, f);
        fread((char *)&start_val, 8, 1, f);
        fread((char *)&comp_type, 4, 1, f);		
	}
    else {
        cnt = ((unsigned int*)host)[0];
		orig_recCount = ((unsigned int*)host + cnt*2)[7];
		bits = ((unsigned int*)host + cnt*2)[8];
		orig_lower_val = ((long long int*)((unsigned int*)host + cnt*2 + 9))[0];
		fit_count = ((unsigned int*)host + cnt*2)[11];
		start_val = ((long long int*)((unsigned int*)host + cnt*2 + 12))[0];
		comp_type = ((unsigned int*)host + cnt*2)[14];	  
    };
	*mRecCount = orig_recCount;

	//cout << "Decomp Header " << orig_recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << endl;  
  
    thrust::device_ptr<unsigned long long int> decomp = thrust::device_malloc<unsigned long long int>(cnt);
	
	
    unsigned long long int* raw_decomp = thrust::raw_pointer_cast(decomp);	  
	if(f)
        hipMemcpy( (void*)raw_decomp, host, cnt*8, hipMemcpyHostToDevice);
	else	
	    hipMemcpy( (void*)raw_decomp, (void*)((unsigned int*)host + 5), cnt*8, hipMemcpyHostToDevice);

    void* d_v;
    hipMalloc((void **) &d_v, 12);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
	
    void* s_v;
    hipMalloc((void **) &s_v, 8);
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;	
	

    thrust::counting_iterator<unsigned int, thrust::device_system_tag> begin(0);
    if(tp == 0) {
        decompress_functor_int ff1(raw_decomp,(int_type*)destination, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + orig_recCount, ff1);
        if(comp_type == 1) {
            thrust::device_ptr<int_type> d_int((int_type*)destination);
            d_int[0] = start_val;			
            thrust::inclusive_scan(d_int, d_int + orig_recCount, d_int);
        };
    }
    else {
	
        decompress_functor_float ff1(raw_decomp,(long long int*)destination, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + orig_recCount, ff1);	
        if(comp_type == 1) {
            thrust::device_ptr<long long int> d_int((long long int*)destination);
            d_int[0] = start_val;
            thrust::inclusive_scan(d_int, d_int + orig_recCount, d_int);
        };

    };
    hipFree(d_v);
    hipFree(s_v);
    thrust::device_free(decomp);
    return 1;

}

unsigned long long int pfor_delta_compress(void* source, unsigned int source_len, char* file_name, void*& host, bool tp, unsigned long long int sz)
{
    long long int orig_lower_val, orig_upper_val, start_val, real_lower, real_upper;
    unsigned int  bits, recCount;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 1; // FOR-DELTA
    
    if(tp == 0)
        recCount = source_len/int_size;
    else
        recCount = source_len/float_size;
		
    void* ss;
    CUDA_SAFE_CALL(hipMalloc((void **) &ss, recCount*float_size));

    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
        thrust::device_ptr<int_type> d_ss((int_type*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);	

        start_val = d_ss[0];
		if(recCount > 1)
            d_ss[0] = d_ss[1];
		
		orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
		orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));	

		real_lower = s[0];
		real_upper = s[recCount-1];
		//cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
	    //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " <<  orig_upper_val-orig_lower_val << " values " << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
		if (bits == 0)
		    bits = 1;

    }
    else {
        thrust::device_ptr<long long int> s((long long int*)source);
        thrust::device_ptr<long long int> d_ss((long long int*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);
        start_val = d_ss[0];
		if(recCount > 1)
            d_ss[0] = d_ss[1];

		orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
		orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));	
		real_lower = s[0];
		real_upper = s[recCount-1];
		
		//cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
	    //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " << orig_upper_val-orig_lower_val << " values" << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
		if (bits == 0)
		    bits = 1;		
    };

    thrust::counting_iterator<unsigned int, thrust::device_system_tag> begin(0);

    fit_count = bit_count/bits;
    void* d_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);

    void* s_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;


    //void* d;
    //CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));

    thrust::device_ptr<char> dd((char*)source);
    thrust::fill(dd, dd+source_len,0);

	//cout << "FF " << orig_lower_val << " " << bits << " " << fit_count << " " << bit_count << endl;

    if (tp == 0) {
        compress_functor_int ff((int_type*)ss,(unsigned long long int*)source, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)ss,(unsigned long long int*)source, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + recCount, ff);
    };
	

    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)source);
	
    // make an addition  sequence

	thrust::device_ptr<unsigned long long int> add_seq((unsigned long long int*)ss);
    thrust::constant_iterator<unsigned long long int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned long long int>());

    unsigned int cnt = (recCount)/fit_count;
    if (recCount%fit_count > 0)
        cnt++;
    thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

	//for(int i = 0; i < 10;i++)
	//  cout << "FIN " << fin_seq[i] << endl;

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);
	 
	if(file_name) { 
        hipMemcpy( host, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        fstream binary_file(file_name,ios::out|ios::binary|ios::app);
        binary_file.write((char *)&cnt, 4);
		binary_file.write((char *)&real_lower, 8);
		binary_file.write((char *)&real_upper, 8);
        binary_file.write((char *)host,cnt*8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&recCount, 4);
        binary_file.write((char *)&bits, 4);        
		binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&fit_count, 4);
        binary_file.write((char *)&start_val, 8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.close();
	}	
    else {
		char* hh;
    	resize_compressed(host, sz, cnt*8 + 15*4, 0); 
	    hh = ((char*)host + sz);
		((unsigned int*)hh)[0] = cnt;
		((long long int*)(hh+4))[0] = real_lower;
		((long long int*)(hh+12))[0] = real_upper;
		hipMemcpy( hh + 20, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
		((unsigned int*)hh)[5+cnt*2] = comp_type;
		((unsigned int*)hh)[6+cnt*2] = cnt;
		((unsigned int*)hh)[7+cnt*2] = recCount;
		((unsigned int*)hh)[8+cnt*2] = bits;
		((long long int*)((char*)hh+36+cnt*8))[0] = orig_lower_val;
		((unsigned int*)hh)[11+cnt*2] = fit_count;
		((long long int*)((char*)hh+48+cnt*8))[0] = start_val;		
		((unsigned int*)hh)[14+cnt*2] = comp_type;		
    };			

    //thrust::device_free(add_seq);
    thrust::device_free(fin_seq);
    //hipFree(d);
    hipFree(ss);
    hipFree(d_v);
    hipFree(s_v);
    return sz + cnt*8 + 15*4; 
}

unsigned long long int pfor_dict_compress(char** d_columns, unsigned int mColumnCount, char* file_name, unsigned int source_len, void*& host, unsigned long long int sz)
{
    unsigned int comp_type = 2; // DICT
    long long int start_val = 0;
    long long int orig_lower_val;
	
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(source_len);
    thrust::sequence(permutation, permutation+source_len);
    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
	
    void* temp;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, source_len));
	
    for(int j=mColumnCount-1; j>=0 ; j--)
        update_permutation_char(d_columns[j], raw_ptr, source_len, (char*)temp, "ASC");

    for(int j=mColumnCount-1; j>=0 ; j--)
        apply_permutation_char(d_columns[j], raw_ptr, source_len, (char*)temp);
		
    hipFree(temp);
	
// group by the vectors
    bool *grp;
    CUDA_SAFE_CALL(hipMalloc((void **) &grp, source_len * sizeof(bool)));
    thrust::device_ptr<bool> d_grp(grp);
    thrust::sequence(d_grp, d_grp+source_len, 0, 0);
	
    thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(source_len);
    d_group[source_len-1] = 1;

    for(unsigned int j=0; j < mColumnCount; j++) {
        thrust::device_ptr<char> d_col(d_columns[j]);
        thrust::transform(d_col, d_col + source_len - 1, d_col+1, d_group, thrust::not_equal_to<char>());
        thrust::transform(d_group, d_group+source_len, d_grp, d_grp, thrust::logical_or<int>());
    };
	

    thrust::device_free(d_group);
    thrust::device_ptr<unsigned int> d_grp_int = thrust::device_malloc<unsigned int>(source_len);
    thrust::transform(d_grp, d_grp+source_len, d_grp_int, bool_to_int());
    //thrust::device_free(d_grp);
    unsigned int grp_count = thrust::reduce(d_grp_int, d_grp_int+source_len);
	if(grp_count == 1)
	    grp_count++;
		
    //if(grp_count < source_len)
//        cout << "Compressable to " << grp_count << endl;
//    cout << "grp count " << grp_count << endl;

    unsigned int bits = (unsigned int)log2((double)(grp_count))+1;
  
    thrust::device_ptr<int_type> permutation_final = thrust::device_malloc<int_type>(source_len);

    thrust::exclusive_scan(d_grp_int, d_grp_int+source_len, d_grp_int, 0);
    thrust::scatter(d_grp_int, d_grp_int+source_len, permutation, permutation_final);
    thrust::device_free(permutation);
	

//	for(int z = 0; z < 10; z++)
//	cout << "RES " << permutation_final[z] << endl;

    unsigned int fit_count = 64/bits;

    void* d_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);

    void* s_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

    dd_sv[0] = 0;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = 64;

    thrust::counting_iterator<unsigned int, thrust::device_system_tag> begin(0);

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, source_len*float_size));
	
    thrust::device_ptr<char> dd((char*)d);
    thrust::fill(dd, dd+source_len,0);

    compress_functor_int ff(thrust::raw_pointer_cast(permutation_final),(unsigned long long int*)d, (long long int*)s_v, (unsigned int*)d_v);
    thrust::for_each(begin, begin + source_len, ff);

    hipFree(d_v);
    hipFree(s_v);

    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)d);

    // make an addition  sequence
    //thrust::device_ptr<unsigned long long int> add_seq = thrust::device_malloc<unsigned long long int>(source_len);
    thrust::constant_iterator< long long int> iter(fit_count);
    thrust::sequence(permutation_final, permutation_final + source_len, 0, 1);
    thrust::transform(permutation_final, permutation_final + source_len, iter, permutation_final, thrust::divides<long long int>());
	

    unsigned int cnt = (source_len)/fit_count;
    if (source_len%fit_count > 0)
        cnt++;
    thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);

    //cout << "fin seq " << cnt << " " << source_len <<  endl;

    thrust::reduce_by_key(permutation_final, permutation_final+source_len,s_copy1,thrust::make_discard_iterator(), fin_seq);
    //thrust::device_free(add_seq);
    orig_lower_val = 0;
	
	
	if (file_name) {
        hipMemcpy( host, (void *)thrust::raw_pointer_cast(fin_seq), cnt*8, hipMemcpyDeviceToHost);
		thrust::device_free(fin_seq);
        fstream binary_file(file_name,ios::out|ios::binary|ios::app);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)host,cnt*8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&source_len, 4);

    // write a dictionary
        binary_file.write((char *)&grp_count, 4);
    // create dictionary
        thrust::device_ptr<char> dict = thrust::device_malloc<char>(grp_count);
        for(unsigned int j=0; j < mColumnCount; j++) {
            thrust::device_ptr<char> d_col(d_columns[j]);
            thrust::transform(d_grp, d_grp+source_len, d_grp_int, bool_to_int());
            thrust::copy_if(d_col,d_col+source_len,d_grp_int, dict, nz<unsigned int>());
            hipMemcpy( host, (void *)thrust::raw_pointer_cast(dict), grp_count, hipMemcpyDeviceToHost);
            binary_file.write((char *)host,grp_count);
        };
        thrust::device_free(dict);
        binary_file.write((char *)&grp_count, 4);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&source_len, 4);
        binary_file.write((char *)&bits, 4);
        binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&fit_count, 4);
        binary_file.write((char *)&start_val, 8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.close();
	}	
	else {
        char* hh;
	    resize_compressed(host, sz, cnt*8 + mColumnCount*grp_count + 14*4, 0); 
	    hh = ((char*)host + sz);	
		((unsigned int*)hh)[0] = cnt;
		hipMemcpy( (unsigned int*)hh + 1, (void *)thrust::raw_pointer_cast(fin_seq), cnt*8, hipMemcpyDeviceToHost);	
		thrust::device_free(fin_seq);
		((unsigned int*)hh)[1+cnt*2] = comp_type;
		((unsigned int*)hh)[2+cnt*2] = source_len;
        // write a dictionary
		((unsigned int*)hh)[3+cnt*2] = grp_count;
        // create dictionary
        thrust::device_ptr<char> dict = thrust::device_malloc<char>(grp_count);
        for(unsigned int j=0; j < mColumnCount; j++) {
            thrust::device_ptr<char> d_col(d_columns[j]);
            thrust::transform(d_grp, d_grp+source_len, d_grp_int, bool_to_int());
            thrust::copy_if(d_col,d_col+source_len,d_grp_int, dict, nz<unsigned int>());
            hipMemcpy( (void*)(hh+16+cnt*8+j*grp_count), (void *)thrust::raw_pointer_cast(dict), grp_count, hipMemcpyDeviceToHost);            
        };
		thrust::device_free(dict);
		((unsigned int*)(hh+16+cnt*8+mColumnCount*grp_count))[0] = grp_count;
		((unsigned int*)(hh+20+cnt*8+mColumnCount*grp_count))[0] = cnt;
		((unsigned int*)(hh+24+cnt*8+mColumnCount*grp_count))[0] = source_len;
		((unsigned int*)(hh+28+cnt*8+mColumnCount*grp_count))[0] = bits;     
    	((long long int*)(hh+32+cnt*8+mColumnCount*grp_count))[0] = orig_lower_val;    
		((unsigned int*)(hh+40+cnt*8+mColumnCount*grp_count))[0] = fit_count;    
		((long long int*)(hh+44+cnt*8+mColumnCount*grp_count))[0] = start_val;    	
		((unsigned int*)(hh+52+cnt*8+mColumnCount*grp_count))[0] = comp_type;    	
    };		
	
  
    thrust::device_free(permutation_final);
    thrust::device_free(d_grp_int);
    hipFree(d);
    thrust::device_free(d_grp);   
	
	return sz + cnt*8 + mColumnCount*grp_count + 14*4;		
	
}



unsigned long long int pfor_compress(void* source, unsigned int source_len, char* file_name, void*& host, bool tp, unsigned long long int sz)
{
    unsigned int recCount;
    long long int orig_lower_val;
    long long int orig_upper_val;
    unsigned int  bits;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 0; // FOR
    long long int start_val = 0;
    bool sorted = 0;
	
    if(tp == 0)
        recCount = source_len/int_size;
    else
        recCount = source_len/float_size;


    // check if sorted
	
    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
        sorted = thrust::is_sorted(s, s+recCount);
    }
    else {
        thrust::device_ptr<long long int> s((long long int*)source);
        sorted = thrust::is_sorted(s, s+recCount);
    };

	
    if(sorted) 
        return pfor_delta_compress(source, source_len, file_name, host, tp, sz);
		

// sort the sequence

    
    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
		
		orig_lower_val = *(thrust::min_element(s, s + recCount));
		orig_upper_val = *(thrust::max_element(s, s + recCount));
		
        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    }
    else {
        thrust::device_ptr<long long int> s((long long int*)source);

		orig_lower_val = *(thrust::min_element(s, s + recCount));
		orig_upper_val = *(thrust::max_element(s, s + recCount));
		
        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    };	
	
    thrust::counting_iterator<unsigned int, thrust::device_system_tag> begin(0);

    fit_count = bit_count/bits;
    void* d_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);

    void* s_v;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));	
    thrust::device_ptr<char> dd((char*)d);
    thrust::fill(dd, dd+source_len,0);

    if (tp == 0) {
        compress_functor_int ff((int_type*)source,(unsigned long long int*)d, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)source,(unsigned long long int*)d, (long long int*)s_v, (unsigned int*)d_v);
        thrust::for_each(begin, begin + recCount, ff);
    };

	
    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)d);

    // make an addition  sequence
    thrust::device_ptr<unsigned int> add_seq = thrust::device_malloc<unsigned int>(recCount);
    thrust::constant_iterator<unsigned int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned int>());
	
    unsigned int cnt = (recCount)/fit_count;
    if (recCount%fit_count > 0)
        cnt++;

    //thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);
	thrust::device_ptr<unsigned long long int> fin_seq((unsigned long long int*)source);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);	

	if(file_name) {
        hipMemcpy( host, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        fstream binary_file(file_name,ios::out|ios::binary|ios::app);
        binary_file.write((char *)&cnt, 4);
		binary_file.write((char *)&orig_lower_val, 8);
		binary_file.write((char *)&orig_upper_val, 8);		
        binary_file.write((char *)host,cnt*8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&recCount, 4);
        binary_file.write((char *)&bits, 4);
        binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&fit_count, 4);
        binary_file.write((char *)&start_val, 8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.close();
	}
    else {
        char* hh;
	 // resize host to sz + cnt*8 + 15
	    resize_compressed(host, sz, cnt*8 + 15*4, 0); 
	    hh = ((char*)host + sz);	
		((unsigned int*)hh)[0] = cnt;
		((long long int*)(hh+4))[0] = orig_lower_val;
		((long long int*)(hh+12))[0] = orig_upper_val;		
		hipMemcpy( hh + 20, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
		((unsigned int*)hh)[5+cnt*2] = comp_type;
		((unsigned int*)hh)[6+cnt*2] = cnt;
		((unsigned int*)hh)[7+cnt*2] = recCount;
		((unsigned int*)hh)[8+cnt*2] = bits;
		((long long int*)(hh+36+cnt*8))[0] = orig_lower_val;
		((unsigned int*)hh)[11+cnt*2] = fit_count;
		((long long int*)(hh+48+cnt*8))[0] = start_val;
		((unsigned int*)hh)[14+cnt*2] = comp_type;		
    };	

    thrust::device_free(add_seq);
    //thrust::device_free(fin_seq);
    hipFree(d);
    hipFree(d_v);
    hipFree(s_v);
    return sz + cnt*8 + 15*4; 
}
