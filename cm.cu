#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/set_operations.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/adjacent_difference.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <queue>
#include <set>
#include <string>
#include <map>
#include <ctime>
#ifdef _WIN64
#include <process.h>
#endif
#include "cm.h"
#include "atof.h"
#include "itoa.h"
#include "compress.cu"


#ifdef _WIN64
#else
#define _FILE_OFFSET_BITS 64
#endif

#ifdef _WIN64
#define fseeko _fseeki64
#define ftello _ftelli64
#else
#define fseeko fseek
#define ftello ftell
#endif


using namespace std;

unsigned int process_count;
long long int runningRecs = 0;
long long int totalRecs = 0;
bool fact_file_loaded = 0;
bool buffersEmpty = 0;

const double gpu_mem = 0.7;  // amount of gpu memory used to keep the record sets. The rest is used as tmp space

map<string,queue<string> > top_type;
map<string,queue<string> > top_value;
map<string,queue<int_type> > top_nums;
map<string,queue<float_type> > top_nums_f;	


template <typename HeadFlagType>
struct head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !left;
    }
};

struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return !(((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};

struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};

struct float_to_long
{

    __host__ __device__
    long long int operator()(const float_type x)
    {
        if ((long long int)((x+EPSILON)*100.0) > (long long int)(x*100.0))
            return (long long int)((x+EPSILON)*100.0);
        else return (long long int)(x*100.0);


    }
};

struct long_to_float
{
    __host__ __device__
    float_type operator()(const long long int x)
    {
        return (((float_type)x)/100.0);
    }
};




struct comp_bits_functor
{
    const int_type a;

    comp_bits_functor(int_type _a) : a(_a) {}

    __host__ __device__
    unsigned int operator()(const int_type& x, const int_type& y) const {
        return ((x&a) == (y&a));
    }
};

struct cmp_functor
{
    const char * src;
    int_type * output;
    const char * str;
    const unsigned int * len;

    cmp_functor(const char * _src, int_type * _output, const char * _str, const unsigned int * _len):
        src(_src), output(_output), str(_str), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        if(src[i] != 0 && output[i] >= 0 && output[i] < *len ) {
            if ( src[i] == str[(*len-output[i]) - 1])
                output[i]++;
            else
                output[i] = -1;
        };
    }
};


void LoadBuffers(void* file_name);
class CudaSet;
unsigned int getChunkCount(CudaSet* a);
unsigned int findSegmentCount(char* file_name);
CudaSet *th;
bool buffersLoaded;


size_t getFreeMem();
unsigned int getSize(CudaSet* a); 
bool zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a);



class CudaChar
{
public:
    char** h_columns;
    char** d_columns;
    char* compressed;
    unsigned int mColumnCount;
    unsigned int mRecCount;

    CudaChar(unsigned int columnCount, unsigned int Recs)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs);
    }
	
    CudaChar(unsigned int columnCount, unsigned int Recs, bool gpu)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs, gpu);
    }
	
    CudaChar(unsigned int columnCount, unsigned int Recs, bool gpu, long long int compressed_size)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs, gpu, compressed_size);
    }
	
	


    ~CudaChar()
    {
        free();
    }
	
	
	void findMinMax(string& minStr, string& maxStr)
	{	
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(mRecCount);					
        thrust::sequence(permutation, permutation+mRecCount);	
	
        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
        void* temp;
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount));	
	
        for(int j=mColumnCount-1; j>=0 ; j--)
            update_permutation_char(d_columns[j], raw_ptr, mRecCount, (char*)temp, "ASC");			
		
        minStr = "";
		maxStr = "";
		
		for(unsigned int j=0; j<mColumnCount; j++) {
		    thrust::device_ptr<char> cc(d_columns[j]);
		    minStr+=cc[permutation[0]];
		    maxStr+=cc[permutation[mRecCount-1]];
		};
		
		hipFree(temp);
		hipFree(raw_ptr);	
	}
	

    void resize(unsigned int addRecs)
    {
       /* unsigned int old_count = mRecCount;
        mRecCount = mRecCount + addRecs;
        for(unsigned int i=0; i <mColumnCount; i++) {
            char* n;
            hipHostMalloc(&n, mRecCount);
            if(old_count != 0) {
                if (old_count < mRecCount)
                    memcpy(n,h_columns[i],old_count);
                else
                    memcpy(n,h_columns[i],mRecCount);
                hipHostFree(h_columns[i]);
            };
            h_columns[i] = n;
        };
	*/
        mRecCount = mRecCount + addRecs;
        for(unsigned int i=0; i <mColumnCount; i++)
            h_columns[i] = (char *)realloc((void *)h_columns[i], mRecCount);
	
    }

    void allocOnDevice(unsigned int RecordCount)
    {
        for(unsigned int i=0; i <mColumnCount; i++)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_columns[i], RecordCount));
        mRecCount = RecordCount;
    }

    void deAllocOnDevice()
    {
        for(unsigned int i=0; i <mColumnCount; i++) {
            if (d_columns[i]) {
                hipFree(d_columns[i]);
                d_columns[i] = 0;
            }
        };
    };

    void free()
    {
        for(unsigned int i=0; i <mColumnCount; i++) {
            if (d_columns[i]) {
                hipFree(d_columns[i]);
                d_columns[i] = 0;
            };
			if(h_columns[i])
                //hipHostFree(h_columns[i]);
	            delete [] h_columns[i];

        };
        delete [] d_columns;
        delete [] h_columns;
        if (compressed) {
		    hipHostFree(compressed);
			compressed = 0;
		};			
    };

    void CopyToGpu(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++)
            hipMemcpy((void *) d_columns[i], (void *) (h_columns[i] + offset), count, hipMemcpyHostToDevice);
    };

    void CopyToHost(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++)
            hipMemcpy((void *) (h_columns[i] + offset), (void *) d_columns[i], count, hipMemcpyDeviceToHost);               			
    };

    int_type* findStr(string str)
    {
        // return a boolean vector of size mRecCount
        thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(mRecCount);
        thrust::sequence(res, res+mRecCount, 1, 0);

        thrust::device_ptr<int_type> v = thrust::device_malloc<int_type>(mRecCount);

        for(int i =0; i < str.length(); i++) {
            thrust::device_ptr<char> dev_ptr(d_columns[i]);
            thrust::device_ptr<char> c = thrust::device_malloc<char>(mRecCount);

            thrust::sequence(c, c+mRecCount, (int)str[i], 0);

            thrust::transform(dev_ptr, dev_ptr+mRecCount, c, v, thrust::equal_to<char>());
            thrust::transform(v, v+mRecCount, res, res, thrust::logical_and<int_type>());
        };
        return thrust::raw_pointer_cast(res);
    };


    bool* cmpStr(string str)
    {

	    if (str[str.size()-1] == '%' && str[0] == '%') { // contains
            if(str.size() > mColumnCount) {
                thrust::device_ptr<bool> res_f = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res_f, res_f+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res_f);
            }
            else {
		
		       return 0;
		
		    };		
		}
        else if(str[str.size()-1] == '%') {  // startsWith

            if(str.size() > mColumnCount) {
                thrust::device_ptr<bool> res_f = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res_f, res_f+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res_f);
            }
            else {

                thrust::device_ptr<bool> v = thrust::device_malloc<bool>(mRecCount);

                str.erase(str.size()-1,1);
                thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res, res+mRecCount, 1, 0);

                for(int i = 0; i < str.size()-1; i++) {
                    thrust::device_ptr<char> dev_ptr(d_columns[i]);
                    thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::constant_iterator<char>(str[i]), v, thrust::equal_to<char>());
                    thrust::transform(v, v+mRecCount, res, res, thrust::logical_and<bool>());
                };
                thrust::device_free(v);
                return thrust::raw_pointer_cast(res);
            };

        }
        else if(str[0] == '%' ) {  // endsWith

            str.erase(0,1);
            thrust::device_ptr<char> dev_str = thrust::device_malloc<char>(str.size());
            thrust::device_ptr<unsigned int> len = thrust::device_malloc<unsigned int>(1);
            thrust::device_ptr<int_type> output = thrust::device_malloc<int_type>(mRecCount);
			thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
            thrust::sequence(output, output+mRecCount, 0, 0);

            len[0] = str.size();
            for(int z=0; z < str.size(); z++)
                dev_str[z] = str[z];

            for(int i = mColumnCount-1; i >= 0; i--) {
                thrust::device_ptr<char> dev_ptr(d_columns[i]);

                thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);

                cmp_functor ff(thrust::raw_pointer_cast(dev_ptr),
                               thrust::raw_pointer_cast(output),
                               thrust::raw_pointer_cast(dev_str),
                               thrust::raw_pointer_cast(len));


                thrust::for_each(begin, begin + mRecCount, ff);


            };
            thrust::transform(output, output+mRecCount, res, to_zero());
            return thrust::raw_pointer_cast(res);
        }		
        else {                          // equal

            thrust::device_ptr<bool> v = thrust::device_malloc<bool>(mRecCount);
            thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
            thrust::sequence(res, res+mRecCount, 1, 0);

            if(mColumnCount < str.length()) 
            {
                thrust::sequence(res, res+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res);
            };

            for(int i = 0; i < mColumnCount; i++) {
                thrust::device_ptr<char> dev_ptr(d_columns[i]);
                if (str.length() >= i+1) 
                    thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::constant_iterator<char>(str[i]), v, thrust::equal_to<char>());
                else
                    thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::constant_iterator<char>(0), v, thrust::equal_to<char>());
                thrust::transform(v, v+mRecCount, res, res, thrust::logical_and<int_type>());
            };
            thrust::device_free(v);
            return thrust::raw_pointer_cast(res);
        };
    };




protected: // methods

    void initialize(unsigned int columnCount, unsigned int Recs)
    {
        //hipError_t errVal;
		
        compressed = 0;

        mColumnCount = columnCount;
        mRecCount = Recs;
        h_columns = new char*[mColumnCount];
        d_columns = new char*[mColumnCount];


        for(unsigned int i=0; i <mColumnCount; i++) {
            //errVal = hipHostMalloc(&h_columns[i], Recs);
            h_columns[i] = new char[Recs];

          /*  if(errVal != hipSuccess) {
                cout << "ALLOC ERROR " << hipGetErrorString(errVal) << endl;
                exit(-1);
            };
		*/	
            d_columns[i] = 0;
        };

    };
	
    void initialize(unsigned int columnCount, unsigned int Recs, bool gpu)
    {
        compressed = 0;
        mColumnCount = columnCount;
        mRecCount = Recs;
        h_columns = new char*[mColumnCount];
        d_columns = new char*[mColumnCount];

        for(unsigned int i=0; i <mColumnCount; i++) {
            d_columns[i] = 0;
			h_columns[i] = 0;
        };
    };
	
    void initialize(unsigned int columnCount, unsigned int Recs, bool gpu, long long int compressed_size)
    {
        mColumnCount = columnCount;
        mRecCount = Recs;
        h_columns = new char*[mColumnCount];
        d_columns = new char*[mColumnCount];

        for(unsigned int i=0; i <mColumnCount; i++) {
            d_columns[i] = 0;
			h_columns[i] = 0;
        };
		hipHostMalloc(&compressed, compressed_size);
		//compressed = new char[compressed_size];
    };	
	

};



class CudaSet
{
public:
    void** h_columns;
    void** d_columns;
    unsigned int mColumnCount;
    unsigned int mRecCount;
    map<string,int> columnNames;
    map<string, FILE*> filePointers;
    bool *grp;
    queue<string> columnGroups;
    bool fact_table; // 1 = fact table, 0 = dimension table
    FILE *file_p;
    unsigned long long int *offsets; // to store the current offsets for compression routines
    unsigned int *seq;
    bool keep;
	map<int,bool> uniqueColumns;
	unsigned int segCount, maxRecs;
	char* name;


    vector< vector<unsigned int> > m_position; //for partition by lower bits
    vector< vector<unsigned int> > m_size;	//for partition by lower bits
    unsigned int m_current;

    unsigned int* type; // 0 - integer, 1-float_type, 2-char
	bool* decimal; // column is decimal - affects only compression
    unsigned int* grp_type; // type of group : SUM, AVG, COUNT etc
    unsigned int* cols; // column positions in a file
    unsigned int grp_count;
    int readyToProcess;
	
    CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
        keep = false;
		offsets = 0;
    }
	
    CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
        keep = false;
		offsets = 0;
    }
	


    CudaSet(unsigned int RecordCount, unsigned int ColumnCount)
    {
        initialize(RecordCount, ColumnCount);
        keep = false;
		offsets = 0;
    };



    CudaSet(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
    {
        initialize(a,b,Recs, op_sel, op_sel_as);
        keep = false;
    };


    ~CudaSet()
    {
        free();
    }
	
	
	
	bool isUnique(unsigned int colIndex) //  run only on already sorted columns
	{	  
	  if (fact_table)
	      uniqueColumns[colIndex] = 0;
	  if (uniqueColumns.find(colIndex) == uniqueColumns.end()) {
          thrust::device_ptr<int_type> d_col((int_type*)d_columns[colIndex]);
		  if(mRecCount == 1 )
		      uniqueColumns[colIndex] = 1;
		  else {	  
		      thrust::device_ptr<unsigned int> d_group = thrust::device_malloc<unsigned int>(mRecCount-1);		  
		  
              thrust::transform(d_col, d_col + mRecCount - 1, d_col+1, d_group, thrust::not_equal_to<int_type>());
		      unsigned int grp_count = thrust::reduce(d_group, d_group+mRecCount-1);		      
		      if(grp_count == mRecCount-1)
		          uniqueColumns[colIndex] = 1;
		      else
		          uniqueColumns[colIndex] = 0;		  
		  };		  

	  };	   
	  return uniqueColumns[colIndex];	
	};
	

    void resize(unsigned int addRecs)
    {
        unsigned int old_count = mRecCount;
        mRecCount = mRecCount + addRecs;
        char* n;
		
		if(!fact_table) {
		    for(unsigned int i=0; i <mColumnCount; i++) {
			    if(type[i] !=2)
                    h_columns[i] = realloc((void *)h_columns[i], mRecCount);
				else 
				((CudaChar*)h_columns[i])->resize(addRecs);				
			};	
		}
        else		

        for(unsigned int i=0; i < mColumnCount; i++) {
            if (type[i] == 0) {
                hipError_t errVal = hipHostMalloc(&n, mRecCount*int_size);
                if(errVal != hipSuccess) {
                    cout << "ALLOC ERROR " << hipGetErrorString(errVal) << endl;
                    exit(0);
                };
                if (old_count != 0) {
                    if (old_count < mRecCount)
                        memcpy(n,h_columns[i],old_count*int_size);
                    else
                        memcpy(n,h_columns[i],mRecCount*int_size);
                    hipHostFree(h_columns[i]);
                };
                h_columns[i] = n;
            }
            else if (type[i] == 1) {                
                hipError_t errVal = hipHostMalloc(&n, mRecCount*float_size);
                if(errVal != hipSuccess) {
                    cout << "ALLOC ERROR " << hipGetErrorString(errVal) << endl;
                    exit(0);
                };				
                if (old_count != 0) {
                    if (old_count < mRecCount)
                        memcpy(n,h_columns[i],old_count*float_size);
                    else
                        memcpy(n,h_columns[i],mRecCount*float_size);
                    hipHostFree(h_columns[i]);
                };
                h_columns[i] = n;
            }
            else
                ((CudaChar*)h_columns[i])->resize(addRecs);
        };

    }


    void allocColumnOnDevice(unsigned int colIndex, unsigned int RecordCount)
    {
        if (type[colIndex] == 0)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_columns[colIndex], RecordCount*int_size));
        else if (type[colIndex] == 1)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_columns[colIndex], RecordCount*float_size));
        else {
            int a = 1;
            ((CudaChar*)h_columns[colIndex])->allocOnDevice(RecordCount);
            d_columns[colIndex] = &a;
        };
    };


    void deAllocColumnOnDevice(unsigned int colIndex)
    {
        if (type[colIndex] == 0 || type[colIndex] == 1) {
            if (d_columns[colIndex]) {
                hipFree(d_columns[colIndex]);
                d_columns[colIndex] = 0;
            };
        }
        else {
            ((CudaChar*)h_columns[colIndex])->deAllocOnDevice();
            d_columns[colIndex] = 0;
        };
    };
	
	void setTypes(CudaSet* b)
	{
        for(unsigned int i=0; i < b->mColumnCount; i++) 
            type[i] = b->type[i];	   
	
	};
	
    void allocOnDevice(unsigned int RecordCount)
    {
        int a = 1;
        for(unsigned int i=0; i < mColumnCount; i++) {
            if (type[i] == 0)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_columns[i], RecordCount*int_size));
            else if (type[i] == 1)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_columns[i], RecordCount*float_size));
            else {
                ((CudaChar*)h_columns[i])->allocOnDevice(RecordCount);
                d_columns[i] = &a;
            };
        };
    };

    void deAllocOnDevice()
    {
        for(unsigned int i=0; i <mColumnCount; i++)
            if (type[i] == 0 || type[i] == 1) {
                hipFree(d_columns[i]);
                d_columns[i] = 0;
            }
            else {
                ((CudaChar*)h_columns[i])->deAllocOnDevice();
                d_columns[i] = 0;
            }
        if(!columnGroups.empty() && mRecCount !=0)
            hipFree(grp);

    };
	
	void resizeDeviceColumn(unsigned int RecCount, unsigned int colIndex)
	{
	    void* d;
        int a = 1;		
		
	    if (RecCount) { 
            if (type[colIndex] == 0) {
                CUDA_SAFE_CALL(hipMalloc( &d, (mRecCount+RecCount)*int_size));
		        if(mRecCount) {
			        hipMemcpy(d, d_columns[colIndex], mRecCount*int_size, hipMemcpyDeviceToDevice);
				    hipFree(d_columns[colIndex]);
				};	
				d_columns[colIndex] = d;
			}	
            else if (type[colIndex] == 1) {
                CUDA_SAFE_CALL(hipMalloc( &d, (mRecCount+RecCount)*float_size));					
    			if(mRecCount) {
		            hipMemcpy(d, d_columns[colIndex], mRecCount*float_size, hipMemcpyDeviceToDevice);
				    hipFree(d_columns[colIndex]);
				};	
				d_columns[colIndex] = d;
			}	
            else {
			    CudaChar *c = (CudaChar*)h_columns[colIndex];
				for(unsigned int j = 0; j < c->mColumnCount; j++) { 
                    CUDA_SAFE_CALL(hipMalloc( &d, mRecCount+RecCount));
					if(mRecCount) {
					    hipMemcpy( d, c->d_columns[j], mRecCount, hipMemcpyDeviceToDevice);
					    hipFree(c->d_columns[j]);
					};	
					c->d_columns[j] = (char*)d;
				};	
                d_columns[colIndex] = &a;
            };
	    };
	};
	
	
	
	void resizeDevice(unsigned int RecCount)
	{
	    if (RecCount)  
	        for(unsigned int i=0; i < mColumnCount; i++) 
			    resizeDeviceColumn(RecCount, i);
	};
	


    int_type copy_filter(CudaSet* b, bool* v, bool del_source)
    {

        thrust::device_ptr<bool> dev_ptr(v);		
        thrust::device_ptr<unsigned int> d_grp_int = thrust::device_malloc<unsigned int>(mRecCount);
        thrust::transform(dev_ptr, dev_ptr+mRecCount, d_grp_int, bool_to_int());
        unsigned int newRecCount = thrust::reduce(d_grp_int, d_grp_int+mRecCount);		
		
        thrust::device_free(d_grp_int);		
        if(b->maxRecs < newRecCount)	
		    b->maxRecs = newRecCount;
			
		//cout << "copy filter " << newRecCount << endl;	

		if (!fact_table) {
			
            void* d;
            CUDA_SAFE_CALL(hipMalloc((void **) &d, newRecCount*float_size));

            for(unsigned int i=0; i < mColumnCount; i++) {
			    //cout << "copy_filter " << i << " " << b->offsets[i] << " " << newRecCount << " " << type[i] << endl;
                if (type[i] == 0 ) {
                    thrust::device_ptr<int_type> src((int_type*)(d_columns)[i]);
                    thrust::device_ptr<int_type> dest((int_type*)d);
                    thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());					
                    b->offsets[i] = pfor_compress(d, newRecCount*int_size, NULL, b->h_columns[i], 0, b->offsets[i]);                     					
                }
                else if (type[i] == 1 ) {
                    thrust::device_ptr<float_type> src((float_type*)(d_columns)[i]);
                    thrust::device_ptr<float_type> dest((float_type*)d);
                    thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());	
					if(!decimal[i]) {
					    // need to allocate b->h_columns[i];
						if(!b->h_columns[i])
                            //hipHostMalloc(&(b->h_columns[i]), newRecCount*float_size);
							b->h_columns[i] = new float_type[newRecCount]; 			
						else {
                            b->resize(newRecCount);							
							b->mRecCount = b->mRecCount - newRecCount;
						};	
                        hipMemcpy((void *) ((float_type*)b->h_columns[i] + b->mRecCount), d, newRecCount*float_size, hipMemcpyDeviceToHost);
					}	
					else {
                        thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                        thrust::transform(dest,dest+newRecCount,d_col_dec, float_to_long());
                        b->offsets[i] = pfor_compress( d, newRecCount*int_size, NULL, b->h_columns[i], 1, b->offsets[i]);
                    };					
                }
                else { //CudaChar
                    CudaChar *s = (CudaChar*)(h_columns)[i];
                    CudaChar *s1 = (CudaChar*)(b->h_columns)[i];
					void *cmp = (void*)s1->compressed;
                    thrust::device_ptr<char> dest((char*)d);					
					
                    for(unsigned int j=0; j < s->mColumnCount; j++) {
                        thrust::device_ptr<char> src(s->d_columns[j]);
                        thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());					
     					hipMemcpy(s->d_columns[j], d, newRecCount, hipMemcpyDeviceToDevice);
					};
                    b->offsets[i] = pfor_dict_compress(s->d_columns, s->mColumnCount, NULL, newRecCount, cmp, b->offsets[i]);
					s1->compressed = (char*)cmp;					
                };
            };
            b->mRecCount = b->mRecCount + newRecCount;
            hipFree(d);		   
		}		
        else {
		    if(!del_source)
		        b->resizeDevice(newRecCount);
            //b->allocOnDevice(newRecCount);
            for(unsigned int i=0; i < mColumnCount; i++) {
			
			    if(del_source) 
				    b->resizeDeviceColumn(newRecCount,i);
			
                if (type[i] == 0 ) {
                    thrust::device_ptr<int_type> src((int_type*)(d_columns)[i]);
                    thrust::device_ptr<int_type> dest((int_type*)b->d_columns[i]);
                    thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());
                }
                else if (type[i] == 1 ) {
                    thrust::device_ptr<float_type> src((float_type*)(d_columns)[i]);
                    thrust::device_ptr<float_type> dest((float_type*)b->d_columns[i]);
                    thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());
                }
                else { //CudaChar
                    CudaChar *s = (CudaChar*)(h_columns)[i];
                    CudaChar *s1 = (CudaChar*)(b->h_columns)[i];
                    for(unsigned int j=0; j < s->mColumnCount; j++) {
                        thrust::device_ptr<char> src(s->d_columns[j]);
                        thrust::device_ptr<char> dest(s1->d_columns[j]);
                        thrust::copy_if(src,src+mRecCount,dev_ptr,dest,nz<int_type>());
                    };
                };
			    if(del_source) 
					deAllocColumnOnDevice(i);
				
            };
            b->mRecCount = newRecCount;
        };			
        return newRecCount;
    };



    CudaSet* copyStruct(unsigned int mCount)
    {

        CudaSet* a = new CudaSet(mCount, mColumnCount);        
        a->fact_table = fact_table;

        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) 
            a->columnNames[(*it).first] = (*it).second;

        for(unsigned int i=0; i < mColumnCount; i++) {

            a->cols[i] = cols[i];
            a->type[i] = type[i];

            if (type[i] == 0) {
			    if(a->fact_table)
                    hipHostMalloc(&(a->h_columns[i]), mCount*int_size);
				else	
				    a->h_columns[i] = new int_type[mCount]; 
			}		
            else if (type[i] == 1) {
			    if(a->fact_table)
                    hipHostMalloc(&(a->h_columns[i]), mCount*float_size);
				else	
				    a->h_columns[i] = new float_type[mCount]; 					
			}	
            else
                a->h_columns[i] = new CudaChar(((CudaChar*) (h_columns)[i])->mColumnCount, mCount);
        };
        return a;
    }
	
    CudaSet* copyDeviceStruct()
    {

        CudaSet* a = new CudaSet(mRecCount, mColumnCount);        
        a->fact_table = fact_table;
		a->segCount = segCount;

        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) 
            a->columnNames[(*it).first] = (*it).second;

        for(unsigned int i=0; i < mColumnCount; i++) {
            a->cols[i] = cols[i];
            a->type[i] = type[i];
        };

		for(unsigned int i=0; i < mColumnCount; i++) {
		    if (type[i] == 2)
			    a->h_columns[i] = new CudaChar(((CudaChar*) (h_columns)[i])->mColumnCount, mRecCount, 1);			
		};	
	    if(!a->fact_table) {
		    a->offsets = new unsigned long long int[mColumnCount];
			for(int i =0; i < mColumnCount;i++)
			   a->offsets[i] = 0; 
	    };

		a->mRecCount = 0;
        return a;
    }
	
	



    void gather(CudaSet* a, CudaSet* b , thrust::device_vector<unsigned int>& d_res1,
                thrust::device_vector<unsigned int>& d_res2, unsigned int segment, queue<string> op_sel)
    {
        int_type RecCount = d_res1.size();
        bool alloc;
        map<string,int>::iterator it;
        unsigned int index;		
	    resizeDevice(RecCount);

        for(unsigned int i=0; i < mColumnCount; i++) {
            alloc = 0;
            it = a->columnNames.find(op_sel.front());
            if(it !=  a->columnNames.end()) {
                index = it->second;

                if( (a->type[index] < 2 && a->d_columns[index] == 0) ) {
                    a->allocColumnOnDevice(index,a->mRecCount);
                    a->CopyColumnToGpu(index);
                    alloc = 1;
                };				

                if (type[i] == 0 ) {						
                    thrust::device_ptr<int_type> src((int_type*)(a->d_columns)[index]);				
                    thrust::device_ptr<int_type> dest((int_type*)d_columns[i] + mRecCount);
                    thrust::gather(d_res1.begin(), d_res1.end(), src, dest);
                    if (alloc)
                        a->deAllocColumnOnDevice(index);
                }
                else if (type[i] == 1 ) {
                    thrust::device_ptr<float_type> src((float_type*)(a->d_columns)[index]);
                    thrust::device_ptr<float_type> dest((float_type*)d_columns[i] + mRecCount);
                    thrust::gather(d_res1.begin(), d_res1.end(), src, dest);
                    if (alloc)
                        a->deAllocColumnOnDevice(index);
                }
                else { //CudaChar
                    CudaChar *s = (CudaChar*)(a->h_columns)[index];
                    CudaChar *dc = (CudaChar*)h_columns[i];
                    alloc = (s->d_columns[0] == 0);
					
                    if(alloc) {
						a->allocColumnOnDevice(index, a->mRecCount);
						a->CopyColumnToGpu(index);
                    };
					
                    for(unsigned int j=0; j < s->mColumnCount; j++) {
					
                        thrust::device_ptr<char> src(s->d_columns[j]);
                        thrust::device_ptr<char> dest(dc->d_columns[j] + mRecCount);
                        thrust::gather(d_res1.begin(), d_res1.end(), src, dest);		
					};

                    if (alloc) 
              			a->deAllocColumnOnDevice(index);
					
                };
            }
            else {
                it = b->columnNames.find(op_sel.front());
                index = it->second;
	
                if (b->type[index] < 2 && b->d_columns[index] == 0) {
                    b->allocColumnOnDevice(index,b->maxRecs);
                    b->CopyColumnToGpu(index, segment);
                    alloc = 1;
                }
                if (type[i] == 0 ) {
                    thrust::device_ptr<int_type> src((int_type*)(b->d_columns)[index]);
                    thrust::device_ptr<int_type> dest((int_type*)d_columns[i] + mRecCount);
                    thrust::gather(d_res2.begin(), d_res2.end(), src, dest);
                    if (alloc)
                        b->deAllocColumnOnDevice(index);
                }
                else if (type[i] == 1 ) {
                    thrust::device_ptr<float_type> src((float_type*)(b->d_columns)[index]);
                    thrust::device_ptr<float_type> dest((float_type*)d_columns[i] + mRecCount);
                    thrust::gather(d_res2.begin(), d_res2.end(), src, dest);
                    if (alloc)
                        b->deAllocColumnOnDevice(index);
                }
                else { //CudaChar
                    CudaChar *s = (CudaChar*)(b->h_columns)[index];
                    CudaChar *dc = (CudaChar*)h_columns[i];
                    alloc = (s->d_columns[0] == 0);
					
                    if(alloc) {
						b->allocColumnOnDevice(index, b->mRecCount);
						b->CopyColumnToGpu(index, segment);
                    };					

                    for(unsigned int j=0; j < s->mColumnCount; j++) {
                        thrust::device_ptr<char> src(s->d_columns[j]);
                        thrust::device_ptr<char> dest(dc->d_columns[j] + mRecCount);
                        thrust::gather(d_res2.begin(), d_res2.end(), src, dest);
                    };
					
                    if (alloc)
					    b->deAllocColumnOnDevice(index);
					
                };
            };	
            op_sel.pop();
        };
		
		mRecCount = mRecCount + d_res1.size();
    }
	
	
    unsigned long long int readSegments(unsigned int segNum, unsigned int colIndex) // read segNum number of segments and return the offset of the next segment
	{
	  unsigned long long int offset = 0; // offset measured in bytes if checking chars and in 4 byte integers if checking ints and decimals
	  unsigned int grp_count;
	  unsigned int data_len; 
	  	  
	  for(int i = 0; i < segNum; i++) {	      
		  if(type[colIndex] != 2) {		      
		      data_len = ((unsigned int*)h_columns[colIndex] + offset)[0]; 
//			  cout << "data len " << data_len << endl;
	          offset = offset + data_len*2 + 15;		
//              cout << "offset " << 	offset << endl;		  
		  }	  
		  else {
		      //cout << "seg start " << endl;
		      CudaChar* c = (CudaChar*)h_columns[colIndex];
			  //cout << "seg start 1" << endl;
			  data_len = ((unsigned int*)(c->compressed + offset))[0]; 
			  //cout << "seg start 2 " << data_len << endl;
		      grp_count = ((unsigned int*)(c->compressed + offset + 8*data_len + 12))[0];		
              //cout << "readseg " << data_len << " " << grp_count << endl;			  
              offset = offset + data_len*8 + 14*4 + grp_count*c->mColumnCount;		  
		  };	  
	  };	  
      return offset;		  
	}

    void CopyToGpu(unsigned int offset, unsigned int count)
    {
	    if (fact_table) {
            for(unsigned int i = 0; i < mColumnCount; i++) {
                switch(type[i]) {
                case 0 :
                    hipMemcpy((void *) d_columns[i], (void *) ((int_type*)h_columns[i] + offset), count*int_size, hipMemcpyHostToDevice);
                    break;
                case 1 :
                    hipMemcpy((void *) d_columns[i], (void *) ((float_type*)h_columns[i] + offset), count*float_size, hipMemcpyHostToDevice);
                    break;
                default :
                    ((CudaChar*)h_columns[i])->CopyToGpu(offset, count);
                };
            };
		}
        else 
		    for(unsigned int i = 0; i < mColumnCount;i++) 
                CopyColumnToGpu(i,  offset, count);
    }
	
	
unsigned int getSegmentRecCount(unsigned int segment, unsigned int colIndex)
{
    long long int data_offset = readSegments(segment,colIndex);
	if (type[colIndex] != 2) {
	    unsigned int data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 	
	    return ((unsigned int*)h_columns[colIndex] + data_offset)[data_len*2 + 7];		
	}	
	else {
        CudaChar* c = (CudaChar*)h_columns[colIndex];
 	    unsigned int data_len = ((unsigned int*)(c->compressed + data_offset))[0]; 
        return ((unsigned int*)(c->compressed + data_offset))[data_len*2 + 2]; 
    };	
}
	
	
	
    void CopyToGpu(unsigned int segment)
    {
	    if (fact_table) {
            for(unsigned int i = 0; i < mColumnCount; i++) {
                switch(type[i]) {
                case 0 :
                    hipMemcpy((void *) d_columns[i], h_columns[i] , mRecCount*int_size, hipMemcpyHostToDevice);
                    break;
                case 1 :
                    hipMemcpy((void *) d_columns[i], h_columns[i] , mRecCount*float_size, hipMemcpyHostToDevice);
                    break;
                default :
                    ((CudaChar*)h_columns[i])->CopyToGpu(0, mRecCount);
                };
            };
		}
        else 
		    for(unsigned int i = 0; i < mColumnCount;i++) 
                CopyColumnToGpu(i,  segment);
    }
	
	


    void CopyColumnToGpu(unsigned int colIndex,  unsigned int segment)
    {
	    if(fact_table) {			
            switch(type[colIndex]) {
            case 0 :
                hipMemcpy((void *) d_columns[colIndex], h_columns[colIndex], mRecCount*int_size, hipMemcpyHostToDevice);
                break;
            case 1 :
                hipMemcpy((void *) d_columns[colIndex], h_columns[colIndex], mRecCount*float_size, hipMemcpyHostToDevice);
                break;
            default :
                ((CudaChar*)h_columns[colIndex])->CopyToGpu(0, mRecCount);
            };			
    	}			
		else {
		    long long int data_offset = readSegments(segment,colIndex);
		
            switch(type[colIndex]) {
                case 0 :
  				    //data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
                    pfor_decompress(d_columns[colIndex] , (void*)((unsigned int*)h_columns[colIndex] + data_offset), &mRecCount, 0, NULL);
                    break;
                case 1 :
   			        if(decimal[colIndex]) {
					    //data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
					    pfor_decompress( d_columns[colIndex] , (void*)((unsigned int*)h_columns[colIndex] + data_offset), &mRecCount, 0, NULL);
                        thrust::device_ptr<long long int> d_col_int((long long int*)d_columns[colIndex]);
                        thrust::device_ptr<float_type> d_col_float((float_type*)d_columns[colIndex] );
                        thrust::transform(d_col_int,d_col_int+mRecCount,d_col_float, long_to_float());
				    }
                    //else // uncompressed float
                       //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
					   // will have to fix it later so uncompressed data will be written by segments too
                    break;
                default :
                    CudaChar* c = (CudaChar*)h_columns[colIndex];
					unsigned int data_len = ((unsigned int*)(c->compressed + data_offset))[0]; 
					grp_count = ((unsigned int*)(c->compressed + data_offset + data_len*8 + 12))[0];
                    pfor_dict_decompress(c->compressed + data_offset, c->h_columns , c->d_columns, &mRecCount, NULL,0, c->mColumnCount, 0);
            };
		};	
    }



    void CopyColumnToGpu(unsigned int colIndex) // copy all segments
    {
	    if(fact_table) {			
            switch(type[colIndex]) {
            case 0 :
                hipMemcpy((void *) d_columns[colIndex], h_columns[colIndex], mRecCount*int_size, hipMemcpyHostToDevice);
                break;
            case 1 :
                hipMemcpy((void *) d_columns[colIndex], h_columns[colIndex], mRecCount*float_size, hipMemcpyHostToDevice);
                break;
            default :
                ((CudaChar*)h_columns[colIndex])->CopyToGpu(0, mRecCount);
            };			
    	}			
		else {
		    long long int data_offset;
			unsigned int totalRecs = 0;
			
			for(unsigned int i = 0; i < segCount; i++) {			
			
		        data_offset = readSegments(i,colIndex);
                switch(type[colIndex]) {
                    case 0 :					    
  				        //data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
                        pfor_decompress((void*)((int_type*)d_columns[colIndex] + totalRecs), (void*)((unsigned int*)h_columns[colIndex] + data_offset), &mRecCount, 0, NULL);
                        break;
                    case 1 :
   			            if(decimal[colIndex]) {
					        //data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
					        pfor_decompress((void*)((int_type*)d_columns[colIndex] + totalRecs), (void*)((unsigned int*)h_columns[colIndex] + data_offset), &mRecCount, 0, NULL);							
                            thrust::device_ptr<long long int> d_col_int((long long int*)d_columns[colIndex] + totalRecs);							
                            thrust::device_ptr<float_type> d_col_float((float_type*)d_columns[colIndex] + totalRecs);
                            thrust::transform(d_col_int,d_col_int+mRecCount,d_col_float, long_to_float());
				        }
                       // else  uncompressed float
                           //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
					       // will have to fix it later so uncompressed data will be written by segments too
                        break;
                    default :
                        CudaChar* c = (CudaChar*)h_columns[colIndex];
                        pfor_dict_decompress(c->compressed + data_offset, c->h_columns , c->d_columns, &mRecCount, NULL,0, c->mColumnCount, totalRecs);
				};	
                totalRecs = totalRecs + mRecCount;				
            };
			mRecCount = totalRecs;
		};	
    }


	

    void CopyColumnToGpu(unsigned int colIndex,  unsigned int offset, unsigned int count)
    {
        if(m_size.empty()) {
		
		    if(fact_table) {			
                switch(type[colIndex]) {
                case 0 :
                    hipMemcpy((void *) d_columns[colIndex], (void *) ((int_type*)h_columns[colIndex] + offset), count*int_size, hipMemcpyHostToDevice);
                    break;
                case 1 :
                    hipMemcpy((void *) d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                    break;
                default :
                    ((CudaChar*)h_columns[colIndex])->CopyToGpu(offset, count);
                };			
			}			
			else {
		        
		        unsigned int start_seg, seg_num, grp_count, data_len, mCount;
			    start_seg = offset/segCount; // starting segment
			    seg_num = count/segCount;    // number of segments that we need
			    long long int data_offset = readSegments(start_seg,colIndex);
				
		
                switch(type[colIndex]) {
                case 0 :
                    for(unsigned int j = 0; j < seg_num; j++) {
						data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
                        pfor_decompress((int_type*)d_columns[colIndex] + segCount*j , (void*)((unsigned int*)h_columns[colIndex] + data_offset), &data_len, 0, NULL);
						data_offset = data_offset + data_len*2 + 15;
					};	

                    break;
                case 1 :
   			        if(decimal[colIndex]) {
						for(unsigned int j = 0; j < seg_num; j++) {
						    data_len = ((unsigned int*)h_columns[colIndex] + data_offset)[0]; 
						    pfor_decompress( (int_type*)d_columns[colIndex] + segCount*j, (void*)((unsigned int*)h_columns[colIndex] + data_offset), &data_len, 0, NULL);
                            thrust::device_ptr<long long int> d_col_int((long long int*)d_columns[colIndex] + segCount*j);
                            thrust::device_ptr<float_type> d_col_float((float_type*)d_columns[colIndex] + segCount*j);
                            thrust::transform(d_col_int,d_col_int+mRecCount,d_col_float, long_to_float());
							data_offset = data_offset + data_len*2 + 15;
                        };
				    }
                    else // uncompressed float
                       hipMemcpy((void *) d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);

                    break;
                default :
                    CudaChar* c = (CudaChar*)h_columns[colIndex];
                    for(unsigned int j = 0; j < seg_num; j++) {
					    data_len = ((unsigned int*)(c->compressed + data_offset))[0]; 
						grp_count = ((unsigned int*)(c->compressed + data_offset + data_len*8 + 12))[0];				
                        pfor_dict_decompress(c->compressed + data_offset, c->h_columns , c->d_columns, &mCount, NULL,0, c->mColumnCount, segCount*j);
						data_offset = data_offset + data_len*8 + 14*4 + grp_count*c->mColumnCount;		  
					};	
                };
            };
        }
        else {
            unsigned int curr_pos = 0;
            for(int i = 0; i < (m_position[0]).size(); i++) {

                switch(type[colIndex]) {
                case 0 :
                    hipMemcpy((void *) ((int_type*)d_columns[colIndex] + curr_pos), (void *) ((int_type*)h_columns[colIndex] + (m_position[m_current])[i]), (m_size[m_current])[i]*int_size, hipMemcpyHostToDevice);
                    break;
                case 1 :
                    hipMemcpy((void *) ((float_type*)d_columns[colIndex]  + curr_pos), (void *) ((float_type*)h_columns[colIndex] + (m_position[m_current])[i]), (m_size[m_current])[i]*float_size, hipMemcpyHostToDevice);
                    break;
                default :
                    CudaChar *c = (CudaChar*)h_columns[colIndex];
                    for(unsigned int k = 0; k < c->mColumnCount; k++)
                        hipMemcpy((void *) (c->d_columns[k] + curr_pos), (void *) (c->h_columns[k] + (m_position[m_current])[i]), (m_size[m_current])[i], hipMemcpyHostToDevice);
                };
                curr_pos = curr_pos + (m_size[m_current])[i];
            };
        };
    }

    void CopyColumnToHost(int colIndex, unsigned int offset, unsigned int RecCount)
    {
        if(m_size.empty()) {
		    if(fact_table) {
                switch(type[colIndex]) {
                case 0 :
                    hipMemcpy((void*)((int_type*)h_columns[colIndex] + offset), d_columns[colIndex], RecCount*int_size, hipMemcpyDeviceToHost);
                    break;
                case 1 :
                    hipMemcpy((void*)((float_type*)h_columns[colIndex] + offset), d_columns[colIndex] , RecCount*float_size, hipMemcpyDeviceToHost);
                    break;
                default :
                    ((CudaChar*)h_columns[colIndex])->CopyToHost(offset,RecCount);
               }
			}   
			else { 
			    unsigned long long int comp_offset = 0;
			    //for(unsigned int i = 0; i < segCount; i++) {
                    switch(type[colIndex]) {
                    case 0 :			
                        comp_offset = pfor_compress(d_columns[colIndex], RecCount*int_size, NULL, h_columns[colIndex], 0, comp_offset);
						break;
                    case 1 :			
					    if (decimal[colIndex]) {
                            thrust::device_ptr<long long int> d_col_dec((long long int*)(d_columns[colIndex]));
							thrust::device_ptr<float_type> d_col_fl((float_type*)(d_columns[colIndex]));
                            thrust::transform(d_col_fl,d_col_fl+RecCount,d_col_dec, float_to_long());						
                            comp_offset = pfor_compress(d_columns[colIndex], RecCount*float_size, NULL, h_columns[colIndex], 0, comp_offset); 													
						}	
                        else { // add code for float
                        } ;
						break;							
                    default :			
                        CudaChar *s = (CudaChar*)(h_columns)[colIndex];       
					    void *cmp = s->compressed;
                        comp_offset = pfor_dict_compress(s->d_columns, s->mColumnCount, NULL, RecCount, cmp, comp_offset);			
                        s->compressed = (char*)cmp;
                    };
				//};	
 					
            };			
        }
    }


    void CopyColumnToHost(int colIndex)
    {
        if(m_size.empty()) 
		    CopyColumnToHost(colIndex, 0, mRecCount);  
        else {
            unsigned int curr_pos = 0;
            for(int i = 0; i < (m_position[0]).size(); i++) {
                switch(type[colIndex]) {
                case 0 :
                    hipMemcpy((void *) ((int_type*)h_columns[colIndex] + (m_position[m_current])[i]), (void *) ((int_type*)d_columns[colIndex] + curr_pos), (m_size[m_current])[i]*int_size, hipMemcpyDeviceToHost);
                    break;
                case 1 :
                    hipMemcpy((void *) ((float_type*)h_columns[colIndex]  + (m_position[m_current])[i]), (void *) ((float_type*)d_columns[colIndex] + curr_pos), (m_size[m_current])[i]*float_size, hipMemcpyDeviceToHost);
                    break;
                default :
                    CudaChar *c = (CudaChar*)h_columns[colIndex];
                    for(unsigned int k = 0; k < c->mColumnCount; k++)
                        hipMemcpy((void *) (c->h_columns[k] + (m_position[m_current])[i]), (void *) (c->d_columns[k] + curr_pos), (m_size[m_current])[i], hipMemcpyDeviceToHost);
                };
                curr_pos = curr_pos + (m_size[m_current])[i];
            };
        };

    }

    void CopyToHost(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++) 
		    CopyColumnToHost(i, offset, count);
    }

    float_type* get_float_type_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return (float_type*)(d_columns[colIndex]);
    }

    int_type* get_int_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return (int_type*)(d_columns[colIndex]);
    }
	
    float_type* get_host_float_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return (float_type*)(h_columns[colIndex]);
    }

    int_type* get_host_int_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return (int_type*)(h_columns[colIndex]);
    }
	


    void GroupBy(queue<string> columnRef)
    {
        if(!columnGroups.empty())
            hipFree(grp);

        int grpInd;

        CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool))); // d_di is the vector for segmented scans
        thrust::device_ptr<bool> d_grp(grp);

        thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

        thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);
        d_group[mRecCount-1] = 1;

        for(int i = 0; i < columnRef.size(); columnRef.pop()) {
            columnGroups.push(columnRef.front()); // save for future references
            int colIndex = columnNames[columnRef.front()];

            if(d_columns[colIndex] == 0) {
                allocColumnOnDevice(colIndex,mRecCount);
                CopyColumnToGpu(colIndex,  0, mRecCount);
                grpInd = 1;
            }
            else
                grpInd = 0;

            if (type[colIndex] == 0) {  // int_type
                thrust::device_ptr<int_type> d_col((int_type*)d_columns[colIndex]);
                thrust::transform(d_col, d_col + mRecCount - 1, d_col+1, d_group, thrust::not_equal_to<int_type>());
                thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());
            }
            else if (type[colIndex] == 1) {  // float_type
                thrust::device_ptr<float_type> d_col((float_type*)d_columns[colIndex]);
                thrust::transform(d_col, d_col + mRecCount - 1, d_col+1, d_group, f_not_equal_to());
                thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());
            }
            else  {  // CudaChar
                char* i1;
                for(unsigned int j=0; j < ((CudaChar*)(h_columns)[colIndex])->mColumnCount; j++) {
                    i1 = (((CudaChar*)(h_columns)[colIndex])->d_columns[j]);
                    thrust::device_ptr<char> d_col(i1);
                    thrust::transform(d_col, d_col + mRecCount - 1, d_col+1, d_group, thrust::not_equal_to<char>());
                    thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<int>());
                }
            };
            if (grpInd == 1)
                deAllocColumnOnDevice(colIndex);
        };

        thrust::device_free(d_group);
        thrust::device_ptr<unsigned int> d_grp_int = thrust::device_malloc<unsigned int>(mRecCount);
        thrust::transform(d_grp, d_grp+mRecCount, d_grp_int, bool_to_int());

        grp_count = thrust::reduce(d_grp_int, d_grp_int+mRecCount);
        thrust::device_free(d_grp_int);
    }


    void addDeviceColumn(int_type* col, int colIndex, string colName, int_type recCount)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 0;
            allocColumnOnDevice(colIndex, recCount);
        };
        // copy data to d columns
        hipMemcpy((void *) d_columns[colIndex], (void *) col, recCount*int_size, hipMemcpyDeviceToDevice);
        mRecCount = recCount;
    };

    void addDeviceColumn(float_type* col, int colIndex, string colName, int_type recCount)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 1;
            allocColumnOnDevice(colIndex, recCount);
        };
        hipMemcpy((void *) d_columns[colIndex], (void *) col, recCount*float_size, hipMemcpyDeviceToDevice);
        mRecCount = recCount;
    };



    void addHostColumn(int_type* col, int colIndex, string colName, int_type recCount, int_type old_reccount, bool one_line)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 0;
            if (!one_line) {
			    if(!fact_table)
                    h_columns[colIndex] = new int_type[old_reccount];
				else	
                    hipHostMalloc(&h_columns[colIndex], old_reccount*int_size);					
			}	
            else {
			    if(!fact_table)
                    h_columns[colIndex] = new int_type[1];
				else	
                    hipHostMalloc(&h_columns[colIndex], int_size);
			};	

        };

        if (!one_line)
            hipMemcpy((void *) ((int_type*)h_columns[colIndex] + mRecCount), (void *) col, recCount*int_size, hipMemcpyDeviceToHost);
        else {
            thrust::device_ptr<int_type> src(col);
            ((int_type*)h_columns[colIndex])[0] = ((int_type*)h_columns[colIndex])[0] + src[0];
        };
    };

    void addHostColumn(float_type* col, int colIndex, string colName, int_type recCount, int_type old_reccount, bool one_line)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 1;
            if (!one_line) {
			    if(!fact_table)
                    h_columns[colIndex] = new float_type[old_reccount];
				else	
                    hipHostMalloc(&h_columns[colIndex], old_reccount*float_size);
			}	
            else {
			    if(!fact_table)
                    h_columns[colIndex] = new float_type[1];
                else
				    hipHostMalloc(&h_columns[colIndex], float_size);
			};	
        };

        if (!one_line)
            hipMemcpy((void *) ((float_type*)h_columns[colIndex] + mRecCount), (void *) col, recCount*float_size, hipMemcpyDeviceToHost);
        else {
            thrust::device_ptr<float_type> src(col);
            ((float_type*)h_columns[colIndex])[0] = ((float_type*)h_columns[colIndex])[0] + src[0];
        };
    };



// a procedure to partition a cudaset by lower bits of the key columns
// the data are on the host.  results should be on the host too

    int partitionCudaSet(unsigned int keyColumn)
    {
        unsigned int current_pos = 0;
        unsigned int copy_count;
        unsigned int chunkCount = getChunkCount(this)/3;
        //chunkCount = 1000;
        unsigned int pieces = mRecCount/chunkCount;
        if (mRecCount%chunkCount != 0)
            pieces++;
        if(pieces == 0)
            pieces = 1;

        unsigned int sz = 0;

        // find a degree of 2 to create bucketCount
        unsigned int degree = 2;

        while(degree < pieces)
            degree = degree*2;

        int_type power = degree-1;
        //cout << "power " <<  power << endl;

        allocOnDevice(chunkCount);

        m_position.clear();
        m_size.clear();
        for(int i = 0 ; i < degree; i++) {
            m_position.push_back(vector <unsigned int>());
            m_size.push_back(vector <unsigned int>());
        };

        // now lets do it for every chunk of records :
        copy_count = chunkCount;
        for(unsigned int i = 0; i < pieces; i++) {

            // copy all columns to device

            if(i == pieces-1)
                copy_count = mRecCount - chunkCount*i;

            CopyToGpu(i*chunkCount, copy_count);


            // power is the count of lower bits of key column that we need to partition on

            thrust::device_ptr<int_type> sd((int_type*)(d_columns)[keyColumn]);
            thrust::device_ptr<unsigned int> d_l = thrust::device_malloc<unsigned int>(copy_count);

            /*	if (i ==0) {
            	  for(int y =0;y < 10;y++) {
            	    int_type a = sd[y];
            		int_type b = a & power;
            	    cout << "SD " << a << " " << b << endl;

            	  };

            	};
            	*/


            unsigned int *d;
            CUDA_SAFE_CALL(hipMalloc((void **) &d, copy_count * float_size));

            for(int_type ii = 0; ii < degree; ii++) {

                thrust::transform(sd,sd + copy_count, thrust::make_constant_iterator(ii), d_l, comp_bits_functor(power));

                sz = thrust::reduce(d_l, d_l+copy_count);

//	  cout << "SZ for " << ii << " = " << sz << endl;

                if(sz != 0) {
                    m_position[ii].push_back(current_pos);
                    m_size[ii].push_back(sz);

                    for(unsigned int j = 0; j < mColumnCount; j++) {
                        if(type[j] == 0) {
                            thrust::device_ptr<int_type> s((int_type*)(d_columns)[j]);
                            thrust::device_ptr<int_type> ti((int_type*)d);
                            thrust::copy_if(s,s + copy_count, d_l, ti, nz<unsigned int>());
                            hipMemcpy((void *) ((int_type*)h_columns[j] + current_pos) , (void*)thrust::raw_pointer_cast(ti), sz*int_size, hipMemcpyDeviceToHost);
                        }
                        else if (type[j] == 1) {
                            thrust::device_ptr<float_type> s((float_type*)(d_columns)[j]);
                            thrust::device_ptr<float_type> tf((float_type*)d);
                            thrust::copy_if(s,s + copy_count, d_l, tf, nz<unsigned int>());
                            hipMemcpy((void *) ((float_type*)h_columns[j] + current_pos) , (void*)thrust::raw_pointer_cast(tf), sz*float_size, hipMemcpyDeviceToHost);
                        }
                        else {
                            CudaChar* bb = (CudaChar*)h_columns[j];
                            for (unsigned int z = 0; z < bb->mColumnCount; z++) {
                                thrust::device_ptr<char> s((char*)(bb->d_columns)[z]);
                                thrust::device_ptr<char> tc((char*)d);
                                thrust::copy_if(s, s + copy_count, d_l, tc, nz<unsigned int>());
                                hipMemcpy((void *) ((char*)bb->h_columns[z] + current_pos) , (void*)thrust::raw_pointer_cast(tc), sz, hipMemcpyDeviceToHost);
                            };
                        };
                    };
                    current_pos = current_pos + sz;
                }
                else {
                    m_position[ii].push_back(current_pos);
                    m_size[ii].push_back(0);
                };
            };
            thrust::device_free(d_l);
            hipFree(d);
        };
        deAllocOnDevice();
        return pieces;
    }



    void Store(char* file_name, char* sep, int limit, bool binary )
    {
        if (mRecCount == 0)
            return;

        unsigned int mCount;


        if(limit != 0 && limit < mRecCount)
            mCount = limit;
        else
            mCount = mRecCount;
			
		
        if(binary == 0) {

            FILE *file_pr = fopen(file_name, "w");
            if (file_pr  == NULL)
                cout << "Could not open file " << file_name << endl;

            char buffer [33];	
		    if(d_columns[0] != 0) {	
                for(unsigned int i=0; i < mColumnCount; i++) {				
                    if (type[i] == 0) 
                        hipHostMalloc(&h_columns[i], int_size*(mCount+1)); 
                    else if (type[i] == 1) 
                        hipHostMalloc(&h_columns[i], float_size*mCount);
                    else {
				        CudaChar *c = (CudaChar*) h_columns[i];
                        for(unsigned int i=0; i <c->mColumnCount; i++)
                            c->h_columns[i] = new char[mCount];
				    };	
		        };	
				bool ch = 0;
				if(!fact_table) {
				    fact_table = 1;
					ch = 1;
				};	
                CopyToHost(0,mCount);				
				if(ch)
				    fact_table = 0;
		    }
            else {   // compressed on the host
			   if(!fact_table) {
			       allocOnDevice(mCount);
			       for(unsigned int i=0; i < mColumnCount; i++) {				       
                       CopyColumnToGpu(i);					   
					   
                        if (type[i] == 0) 
                            hipHostMalloc(&h_columns[i], int_size*(mCount+1)); 
                        else if (type[i] == 1) 
                            hipHostMalloc(&h_columns[i], float_size*mCount);
                        else {
				            CudaChar *c = (CudaChar*) h_columns[i];
                            for(unsigned int i=0; i <c->mColumnCount; i++)
                            c->h_columns[i] = new char[mCount];
						};	
				    };	
					fact_table = 1;
					CopyToHost(0,mCount);				
					fact_table = 0;
		        };					
            };  		

            for(unsigned int i=0; i < mCount; i++) {
                for(unsigned int j=0; j < mColumnCount; j++) {
                    if (type[j] == 0) {
                        sprintf(buffer, "%lld", ((int_type*)h_columns[j])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else if (type[j] == 1) {
                        sprintf(buffer, "%.2f", ((float_type*)h_columns[j])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else {
                        CudaChar* cc = (CudaChar*)(h_columns)[j];
                        char *buf = new char[(cc->mColumnCount)+1];
                        for(int z=0; z<(cc->mColumnCount); z++)
                            buf[z] = (cc->h_columns[z])[i];
                        buf[cc->mColumnCount] = 0;
                        fputs(buf, file_pr);
                        fputs(sep, file_pr);
						delete [] buf;
                    };
                };
                if (i != mCount -1)
                    fputs("\n",file_pr);
            };
            fclose(file_pr);
        }
        else {
            char str[100];
            char col_pos[3];

            bool in_gpu = false;
            if(d_columns[0] != 0)
                in_gpu = true;

			
	        void* d;
			if(!in_gpu) 
                CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));			

            void* host;
            hipHostMalloc(&host, float_size*mCount);			
			
			for(int i = 0; i< mColumnCount; i++)
              if(type[i] == 2 && !in_gpu ) {			    
				  allocColumnOnDevice(i, mCount);
				  CopyColumnToGpu(i,  0, mCount);
			  };	  
				

            for(int i = 0; i< mColumnCount; i++) {

                strcpy(str, file_name);
                strcat(str,".");
                itoaa(cols[i],col_pos);
                strcat(str,col_pos);
                if(type[i] == 0) {
				    if(!in_gpu) {
                        hipMemcpy(d, (void *) (int_type*)h_columns[i], mCount*int_size, hipMemcpyHostToDevice);		
						pfor_compress( d, mCount*int_size, str, host, 0, 0);
					}	
                    else 
					    pfor_compress( d_columns[i], mCount*int_size, str, host, 0, 0);
				}		
                else if(type[i] == 1) {
				    if(decimal[i]) {
				        if(!in_gpu) {
                            hipMemcpy(d, (void *) (float_type*)h_columns[i], mCount*float_size, hipMemcpyHostToDevice);								
                            thrust::device_ptr<float_type> d_col((float_type*)d);
                            thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                            thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
                            pfor_compress( d, mCount*float_size, str, host, 1, 0);
					    }
                        else {					
				            thrust::device_ptr<float_type> d_col((float_type*)d_columns[i]);
                            thrust::device_ptr<long long int> d_col_dec((long long int*)d_columns[i]);
                            thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
                            pfor_compress( d_columns[i], mCount*float_size, str, host, 1, 0);
					    };	
					}
					else { // do not compress
                        fstream binary_file(str,ios::out|ios::binary|fstream::app);
				        binary_file.write((char *)&mCount, 4);
					    if(in_gpu) {
                            hipMemcpy(host, d_columns[i], mCount*float_size, hipMemcpyDeviceToHost);								
                            binary_file.write((char *)host,mCount*float_size);							
					    }
						else
                           binary_file.write((char *)h_columns[i],mCount*float_size);		
						unsigned int comp_type = 3;
                        binary_file.write((char *)&comp_type, 4);						   
                        binary_file.close();
					};
                }
                else {
                    CudaChar *a = (CudaChar*)h_columns[i];
                    pfor_dict_compress(a->d_columns, a->mColumnCount, str, mCount, host, 0);
                };
            };

			for(int i = 0; i< mColumnCount; i++)
              if(type[i] == 2 && !in_gpu)			    
				  deAllocColumnOnDevice(i);
			
			if(!in_gpu) 
			    hipFree(d);
            hipHostFree(host);

        }
    }

    void LoadBigBinaryFile(char* file_name,  long long int diff)
    {
        char str[100];
        char col_pos[3];


		if (d_columns[0] == 0)
            allocOnDevice(mRecCount);
        th = this;

#ifdef _WIN64
//cout << "start waiting " << endl;
        while(!buffersLoaded);
//std::cout<< "waiting time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
#else
        LoadBuffers((void*)file_name);
#endif

        if(buffersEmpty) {
		    fact_file_loaded = 1;
			mRecCount = 0;
			return;
		};	 
		

        for(int i = 0; i< mColumnCount; i++) {		

            strcpy(str, file_name);
            strcat(str,".");
            itoaa(cols[i],col_pos);
            strcat(str,col_pos);
		
            if (type[i] == 0) 
	            pfor_decompress(d_columns[i], h_columns[i], &mRecCount, 0, filePointers[str]);
	        else if (type[i] == 1)  {			
			    if(decimal[i]) {
                    pfor_decompress(d_columns[i],  h_columns[i], &mRecCount, 0, filePointers[str]);				
                    thrust::device_ptr<long long int> d_col_int((long long int*)d_columns[i]);				
                    thrust::device_ptr<float_type> d_col_float((float_type*)d_columns[i]);				
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_col_float, long_to_float());					
				}
                else  // non compressed float
					CopyColumnToGpu(i,0, mRecCount);
            }
            else {
                CudaChar* a = (CudaChar*)h_columns[i];
                pfor_dict_decompress(a->compressed, a->h_columns, a->d_columns, &mRecCount, filePointers[str], 0, a->mColumnCount, 0);
            };
			
        };
#ifdef _WIN64
        buffersLoaded = 0;
        if (mRecCount != diff)
            _beginthread( LoadBuffers, 0, (void*)file_name );

#endif

    }


	



    void LoadFile(char* file_name, char* sep )
    {
        unsigned int count = 0;
        char line[500];
        int l;
        char* field;
        int current_column = 1;

        FILE *file_ptr = fopen(file_name, "r");
        if (file_ptr  == NULL)
            cout << "Could not open file " << file_name << endl;

        unsigned int *seq = new unsigned int[mColumnCount];
        thrust::sequence(seq, seq+mColumnCount,0,1);
        thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);


        while (fgets(line, 500, file_ptr) != NULL ) {

            current_column = 1;
            field = strtok(line,sep);
			cout << line << endl;

            for(int i = 0; i< mColumnCount; i++) {

                while(cols[i] > current_column) {
                    field = strtok(NULL,sep);
                    current_column++;
                };

                if (type[seq[i]] == 0) {
                    if (strchr(field,'-') == NULL) {
                        ((int_type*)h_columns[seq[i]])[count] = atoi(field);
                    }
                    else {   // handling possible dates
                        strncpy(field+4,field+5,2);
                        strncpy(field+6,field+8,2);
                        field[8] = '\0';
                        ((int_type*)h_columns[seq[i]])[count] = atoi(field);
                    };
                }
                else if (type[seq[i]] == 1)
                    ((float_type*)h_columns[seq[i]])[count] = atoff(field);
                else {
                    l = strlen(field);
                    for(int j =0; j< l; j++)
                        (((CudaChar*)h_columns[seq[i]])->h_columns[j])[count] = field[j];
                    for(int j =l; j< ((CudaChar*)(h_columns)[i])->mColumnCount; j++)
                        (((CudaChar*)h_columns[seq[i]])->h_columns[j])[count] = 0;

                };

            };
            count++;
            if (count == mRecCount)
                resize(process_count);
        };

        //delete [] seq;
        fclose(file_ptr);

		cout << "finished " << count << " " << mRecCount << endl;
        resize(count-mRecCount);
		cout << "resized " << endl;


    }


    int LoadBigFile(char* file_name, char* sep )
    {
        unsigned int count = 0;
        char line[500];
        char* field;
        int current_column = 1;
        int l;

        if (file_p == NULL) 
            file_p = fopen(file_name, "r");
        if (file_p  == NULL)
            cout << "Could not open file " << file_name << endl;

        if (seq == 0) {
            seq = new unsigned int[mColumnCount];
            thrust::sequence(seq, seq+mColumnCount,0,1);
            thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);
        };


        while (count < process_count && fgets(line, 500, file_p) != NULL) {

            current_column = 1;
            field = strtok(line,sep);

            for(int i = 0; i< mColumnCount; i++) {

                while(cols[i] > current_column) {
                    field = strtok(NULL,sep);
                    current_column++;
                };
                if (type[seq[i]] == 0) {
                    if (strchr(field,'-') == NULL) {
                        ((int_type*)h_columns[seq[i]])[count] = atoi(field);
                    }
                    else {   // handling possible dates
                        strncpy(field+4,field+5,2);
                        strncpy(field+6,field+8,2);
                        field[8] = '\0';
                        ((int_type*)h_columns[seq[i]])[count] = atoi(field);
                    };
                }
                else if (type[seq[i]] == 1)
                    ((float_type*)h_columns[seq[i]])[count] = atoff(field);
                else {
                    l = strlen(field);
                    for(int j =0; j< l; j++)
                        (((CudaChar*)h_columns[seq[i]])->h_columns[j])[count] = field[j];
                    for(int j =l; j< ((CudaChar*)(h_columns)[i])->mColumnCount; j++)
                        (((CudaChar*)h_columns[seq[i]])->h_columns[j])[count] = 0;
                };

            };
            count++;
        };

        if (count != mRecCount)
            resize(count-mRecCount);

        if(count < process_count)  {
            fclose(file_p);
            return 1;
        }
        else
            return 0;

    }





    void free()  {
        if(mRecCount !=0) {
            for(unsigned int i=0; i <mColumnCount; i++) {
                if(type[i] != 2) {
                    if(d_columns[i] != 0) {
                        hipFree(d_columns[i]);
                        d_columns[i] = 0;
                    };
                    if (h_columns[i] != 0) {
					    if(fact_table) {
                            hipHostFree(h_columns[i]);
						}	
						else {
                          delete [] h_columns[i];			
						};  
						
					};	
                }
                else 
                    ((CudaChar*)h_columns[i])->free();
            };
        };
        delete [] d_columns;
        delete [] h_columns;
        delete type;
        delete cols;
        if (!seq)
            delete seq;

        if(!columnGroups.empty() && mRecCount !=0)
            hipFree(grp);
    };




    bool* logical_and(bool* column1, bool* column2)
    {
        thrust::device_ptr<bool> dev_ptr1(column1);
        thrust::device_ptr<bool> dev_ptr2(column2);

        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

        thrust::device_free(dev_ptr2);
        return column1;

    }


    bool* logical_or(bool* column1, bool* column2)
    {

        thrust::device_ptr<bool> dev_ptr1(column1);
        thrust::device_ptr<bool> dev_ptr2(column2);

        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());

        thrust::device_free(dev_ptr2);

        return column1;

    }



    bool* compare(int_type s, int_type d, int_type op_type)
    {
        bool res;

        if (op_type == 2) // >
            if(d>s) res = 1;
            else res = 0;
        else if (op_type == 1)  // <
            if(d<s) res = 1;
            else res = 0;
        else if (op_type == 6) // >=
            if(d>=s) res = 1;
            else res = 0;
        else if (op_type == 5)  // <=
            if(d<=s) res = 1;
            else res = 0;
        else if (op_type == 4)// =
            if(d==s) res = 1;
            else res = 0;
        else // !=
            if(d!=s) res = 1;
            else res = 0;

        thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
        thrust::sequence(p, p+mRecCount,res,(bool)0);

        return thrust::raw_pointer_cast(p);
    }


    bool* compare(float_type s, float_type d, int_type op_type)
    {
        bool res;

        if (op_type == 2) // >
            if ((d-s) > EPSILON) res = 1;
            else res = 0;
        else if (op_type == 1)  // <
            if ((s-d) > EPSILON) res = 1;
            else res = 0;
        else if (op_type == 6) // >=
            if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;
        else if (op_type == 5)  // <=
            if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;
        else if (op_type == 4)// =
            if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
            else res = 0;
        else // !=
            if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;

        thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
        thrust::sequence(p, p+mRecCount,res,(bool)0);

        return thrust::raw_pointer_cast(p);


    }


    bool* compare(int_type* column1, int_type d, int_type op_type)
    {
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr(column1);


        if (op_type == 2) // >
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
        else // !=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

        return thrust::raw_pointer_cast(temp);

    }

    bool* compare(float_type* column1, float_type d, int_type op_type)
    {
        thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr(column1);

        if (op_type == 2) // >
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
        else // !=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

        return thrust::raw_pointer_cast(res);

    }


    bool* compare(int_type* column1, int_type* column2, int_type op_type)
    {
        thrust::device_ptr<int_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr2(column2);
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

        return thrust::raw_pointer_cast(temp);

    }

    bool* compare(float_type* column1, float_type* column2, int_type op_type)
    {
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<float_type> dev_ptr2(column2);
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

        return thrust::raw_pointer_cast(temp);

    }


    bool* compare(float_type* column1, int_type* column2, int_type op_type)
    {
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr(column2);
        thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);;
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

        thrust::device_free(dev_ptr2);
        return thrust::raw_pointer_cast(temp);

    }




    float_type* op(int_type* column1, float_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr(column1);

        thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

        thrust::device_ptr<float_type> dev_ptr1(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };

        return thrust::raw_pointer_cast(temp);

    }




    int_type* op(int_type* column1, int_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr2(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
        }
        else  {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
        }

        return thrust::raw_pointer_cast(temp);

    }

    float_type* op(float_type* column1, float_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<float_type> dev_ptr2(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
        };

        return thrust::raw_pointer_cast(temp);

    }

    int_type* op(int_type* column1, int_type d, string op_type, int reverse)
    {
        thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
        thrust::fill(temp, temp+mRecCount, d);

        thrust::device_ptr<int_type> dev_ptr1(column1);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());

        };

        return thrust::raw_pointer_cast(temp);

    }

    float_type* op(int_type* column1, float_type d, string op_type, int reverse)
    {
        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::fill(temp, temp+mRecCount, d);

        thrust::device_ptr<int_type> dev_ptr(column1);
        thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
        thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
        }
        else  {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };

        thrust::device_free(dev_ptr1);
        return thrust::raw_pointer_cast(temp);

    }



    float_type* op(float_type* column1, float_type d, string op_type,int reverse)
    {
        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr1(column1);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
        }
        else	{
            if (op_type.compare("MUL") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };

        return thrust::raw_pointer_cast(temp);

    }


protected: // methods


    void initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name) // compressed data for DIM tables
    {
        mColumnCount = nameRef.size();
        h_columns = new void*[mColumnCount];
        d_columns = new void*[mColumnCount];
        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
		decimal = new bool[mColumnCount];
        file_p = NULL;
		long long int sz;
		FILE* f;
		char f1[100];
		
        readyToProcess = 1;
        mRecCount = Recs;

        for(unsigned int i=0; i < mColumnCount; i++) {

            columnNames[nameRef.front()] = i;
            cols[i] = colsRef.front();
            d_columns[i] = 0;
            seq = 0;
            
            strcpy(f1, file_name);
            strcat(f1,".");
            char col_pos[3];
            itoaa(colsRef.front(),col_pos);
            strcat(f1,col_pos); // read the size of the file			
						
            f = fopen (f1 , "rb" );
			fseeko(f, 0, SEEK_END);
            sz = ftello(f);			
			fseeko(f, 0, SEEK_SET);
			

            if ((typeRef.front()).compare("int") == 0) {
                type[i] = 0;
				decimal[i] = 0;
	            h_columns[i] = new char[sz]; 			

//                if( hipSuccess != hipHostMalloc(&h_columns[i], sz)) {
//                    cout << "couldn't allocate " << sz << " bytes " << endl;
//                    exit(-1);
//				};	
            }				
            else if ((typeRef.front()).compare("float") == 0) {
                type[i] = 1;
				decimal[i] = 0;
				h_columns[i] = new char[sz]; 			
//                if( hipSuccess != hipHostMalloc(&h_columns[i], sz)) {
//                    cout << "couldn't allocate " << sz << " bytes " << endl;
//                    exit(-1);
//				};					
            }
            else if ((typeRef.front()).compare("decimal") == 0) {
                type[i] = 1;
				decimal[i] = 1;
				h_columns[i] = new char[sz]; 			
//                if( hipSuccess != hipHostMalloc(&h_columns[i], sz)) {
//                    cout << "couldn't allocate " << sz << " bytes " << endl;
//                    exit(-1);				
//				};
            }			
            else {
                type[i] = 2;
				decimal[i] = 0;				
                h_columns[i] = new CudaChar(sizeRef.front(), Recs, 0, sz);				
                size_t tt = fread(((CudaChar*)h_columns[i])->compressed,sz,1,f);								
            };
			if(type[i] != 2) 
                fread(h_columns[i],sz,1,f);			
			
 		    fclose(f);										
            nameRef.pop();
            typeRef.pop();
            sizeRef.pop();
            colsRef.pop();			
        };
    };



    void initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
    {
        mColumnCount = nameRef.size();
        h_columns = new void*[mColumnCount];
        d_columns = new void*[mColumnCount];
        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
		decimal = new bool[mColumnCount];
		
        file_p = NULL;

        readyToProcess = 1;
        mRecCount = Recs;
		segCount = 1;

        for(unsigned int i=0; i < mColumnCount; i++) {

            columnNames[nameRef.front()] = i;
            cols[i] = colsRef.front();
            d_columns[i] = 0;
            seq = 0;

            if ((typeRef.front()).compare("int") == 0) {
                type[i] = 0;
				decimal[i] = 0;
                if( hipSuccess != hipHostMalloc(&h_columns[i], int_size*(Recs+1))) {
                    cout << "couldn't allocate " << int_size*(Recs+1) << " bytes " << endl;
                    exit(-1);
                };
            }
            else if ((typeRef.front()).compare("float") == 0) {
                type[i] = 1;
				decimal[i] = 0;
                if( hipSuccess != hipHostMalloc(&h_columns[i], float_size*(Recs+1))) {
                    cout << "couldn't allocate " << float_size*Recs << " bytes " << endl;
                    exit(-1);
                };
            }
            else if ((typeRef.front()).compare("decimal") == 0) {
                type[i] = 1;
				decimal[i] = 1;
                if( hipSuccess != hipHostMalloc(&h_columns[i], float_size*(Recs+1))) {
                    cout << "couldn't allocate " << float_size*Recs << " bytes " << endl;
                    exit(-1);
                };
            }
			
            else {
                type[i] = 2;
				decimal[i] = 0;
                h_columns[i] = new CudaChar(sizeRef.front(), Recs);
            };
            nameRef.pop();
            typeRef.pop();
            sizeRef.pop();
            colsRef.pop();
        };
    };

    void initialize(unsigned int RecordCount, unsigned int ColumnCount)
    {
        mRecCount = RecordCount;
        mColumnCount = ColumnCount;
		
        h_columns = new void*[mColumnCount];
        d_columns = new void*[mColumnCount];

        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
		decimal = new bool[mColumnCount];
		
        seq = 0;

        for(int i =0; i < mColumnCount; i++) {
            d_columns[i] = 0;
            h_columns[i] = 0;
            cols[i] = i;
        };
    };
	

    void initialize(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
    {
        mRecCount = Recs;
        mColumnCount = op_sel.size();

        h_columns = new void*[mColumnCount];
		
        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
		decimal = new bool[mColumnCount];

        map<string,int>::iterator it;
        map<int,string> columnNames1;
        readyToProcess = 1;
		
        if (b->fact_table == 1 || a->fact_table == 1) 
            fact_table = 1;
        else {
            fact_table = 0;
		//    offsets = new unsigned long long int[mColumnCount];		
		//	for(unsigned int i = 0; i < mColumnCount;i++)
		//	    offsets[i]= 0;
        };			
			

        d_columns = new void*[mColumnCount];
        for(int i =0; i < mColumnCount; i++) {
            d_columns[i] = 0;			
			h_columns[i] = 0;
		};	

        seq = 0;
        unsigned int i = 0;

        while(!op_sel_as.empty()) {
            columnNames[op_sel_as.front()] = i;
            op_sel_as.pop();
            i++;
        };

        if (Recs != 0) {
		    unsigned int index;
            for(unsigned int i=0; i < mColumnCount; i++) {

                d_columns[i] = 0;				
                if((it = a->columnNames.find(op_sel.front())) !=  a->columnNames.end()) {
                    index = it->second;
                    cols[i] = i;
					decimal[i] = a->decimal[i];

                    if ((a->type)[index] == 0) 
                        type[i] = 0;
                    else if ((a->type)[index] == 1) 
                        type[i] = 1;
                    else {
                        type[i] = 2;
                        h_columns[i] = new CudaChar(((CudaChar*) (a->h_columns)[index])->mColumnCount, Recs, 1);						
                    };
                }
                else {
                    it = b->columnNames.find(op_sel.front());
                    index = it->second;

                    cols[i] = i;
					decimal[i] = b->decimal[index];

                    if ((b->type)[index] == 0) 
                        type[i] = 0;
                    else if ((b->type)[index] == 1) 
                        type[i] = 1;
                    else {
                        type[i] = 2;
                        h_columns[i] = new CudaChar(((CudaChar*) (b->h_columns)[index])->mColumnCount, Recs, 1);						
                    };
                }
                op_sel.pop();
            };
        };
        //if(Recs != 0)
        //    allocOnDevice(Recs);
	    mRecCount = 0;		
    }

};



int reverse_op(int op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t free, total;

    hipMemGetInfo(&free, &total);
//  cout << "Free memory " << free/(1024 * 1024) << " Mbytes out of " << total/(1024 * 1024) << " Mbytes" << endl;
    return free;
} ;

// a procedure that determines how many records of a recordset can fit into GPU memory

unsigned int getSize(CudaSet* a) 
{
    unsigned int sz = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        switch(a->type[i]) {
        case 0 :
            sz=sz+int_size;
            break;
        case 1 :
            sz=sz+float_size;
            break;
        default :
            sz = sz +((CudaChar*)a->h_columns[i])->mColumnCount;
        };
    };
    return sz;
}

unsigned int getChunkCount(CudaSet* a)
{
    unsigned int sz = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        switch(a->type[i]) {
        case 0 :
            sz=sz+int_size;
            break;
        case 1 :
            sz=sz+float_size;
            break;
        default :
            sz = sz +((CudaChar*)a->h_columns[i])->mColumnCount;
        };
    };
    unsigned int t =  (getFreeMem() *  gpu_mem)/(sz*2);
    if (t > a->mRecCount)
        return a->mRecCount;
    else
        return t;

}

bool joinResInGpu(CudaSet* a, CudaSet* b, unsigned int res_size)
{
    unsigned int sz = 0;
    unsigned int sz1 = 0;
    unsigned int largest_col = 0;
    bool l_a = 1;

    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        switch(a->type[i]) {
        case 0 :
            sz=sz+int_size;
            if(largest_col < int_size)
                largest_col = int_size;
            break;
        case 1 :
            sz=sz+float_size;
            if(largest_col < float_size)
                largest_col = float_size;
            break;
        default :
            sz = sz +((CudaChar*)a->h_columns[i])->mColumnCount;
            if(largest_col < 1)
                largest_col = 1;
        };
    };

    for(unsigned int i = 0; i < b->mColumnCount; i++) {
        switch(b->type[i]) {
        case 0 :
            sz1=sz1+int_size;
            if(largest_col < int_size) {
                largest_col = int_size;
                l_a = 0;
            };
            break;
        case 1 :
            sz1=sz1+float_size;
            if(largest_col < float_size) {
                largest_col = float_size;
                l_a = 0;
            };
            break;
        default :
            sz1 = sz1 +((CudaChar*)b->h_columns[i])->mColumnCount;
            if(largest_col < 1)
                largest_col = 1;
        };
    };

    unsigned int res;
    if(l_a)
        res = (sz+sz1)*res_size + (sz*a->mRecCount);
    else
        res = (sz+sz1)*res_size + (sz1*b->mRecCount);

    if  (getFreeMem() > res)
        return 1;
    else
        return 0;

};





void LoadBuffers(void* file_name)
{
    char str[100];
    char col_pos[3];
    unsigned int cnt;
	long long int lower_val, upper_val;
	map<unsigned int,unsigned int> counts;
	bool check_res = 0;
	FILE* f;

    while(runningRecs < totalRecs && !check_res) {
        for(int i = 0; i< th->mColumnCount; i++) {	
            strcpy(str, (char*)file_name);
            strcat(str,".");
            itoaa(th->cols[i],col_pos);
            strcat(str,col_pos);	

            if (th->filePointers.find(str) == th->filePointers.end())
                th->filePointers[str] = fopen(str, "rb");
            f = th->filePointers[str];
		
            if (th->type[i] == 0 || (th->type[i] == 1 && th->decimal[i]))  {
                fread(&cnt, 4, 1, f);
			    counts[i] = cnt;
			    fread(&lower_val, 8, 1, f);
			    fread(&upper_val, 8, 1, f);
			    //cout << "segment upper lower " << upper_val << " " << lower_val << endl;
			    if (th->type[i] == 0) {
			        ((int_type*)(th->h_columns[i]))[0] = lower_val;
			        ((int_type*)(th->h_columns[i]))[1] = upper_val;
			    }
                else {			
			        ((float_type*)(th->h_columns[i]))[0] = ((float_type)lower_val)/100.0;
			        ((float_type*)(th->h_columns[i]))[1] = ((float_type)upper_val)/100.0;			
			    };
            }		
	    };	
		

	    if(!top_type[th->name].empty()) {
	        check_res = zone_map_check(top_type[th->name],top_value[th->name],top_nums[th->name],top_nums_f[th->name],th);
  	        //cout << "check result "	<< check_res << endl;
		    if (!check_res) {   // do not process segment, move the pointers to the next segment
		        runningRecs = runningRecs + th->maxRecs;
                if (runningRecs >= totalRecs) {
                   buffersEmpty = 1;	
				   buffersLoaded = 1;
				   return;
				}	   
				else   {
			    // adjust file pointers	
				    for(int z = 0; z< th->mColumnCount; z++) {						
					
			            strcpy(str, (char*)file_name);
                        strcat(str,".");
                        itoaa(th->cols[z],col_pos);
                        strcat(str,col_pos);	
						f = th->filePointers[str];

						if (th->type[z] == 0 || (th->type[z] == 1 && th->decimal[z]))  
						    fseeko(f, counts[z]*8 + 40, SEEK_CUR);						
						else if (th->type[z] == 1 && !th->decimal[z]) 
						    fseeko(f, counts[z]*8 + 8, SEEK_CUR);						
						else {
                            unsigned int grp_count;
                            CudaChar *c = (CudaChar*)th->h_columns[z];
                            fread(&cnt, 4, 1, f);
							fseeko(f,cnt*8 + 8,SEEK_CUR);		
							fread(&grp_count, 4, 1, f);
                            fseeko(f,grp_count*c->mColumnCount,SEEK_CUR);		
                        };						
					};
				};
		    };
	    }
		else
		    check_res = 1;
	};	
	
	
    for(int i = 0; i< th->mColumnCount; i++) {	
        strcpy(str, (char*)file_name);
        strcat(str,".");
        itoaa(th->cols[i],col_pos);
        strcat(str,col_pos);	

        f = th->filePointers[str];		

        if (th->type[i] == 0 || (th->type[i] == 1 && th->decimal[i]))  {
            //fread(&cnt, 4, 1, f);
			//fread(&lower_val, 8, 1, f);
			//fread(&upper_val, 8, 1, f);
            fread(th->h_columns[i],counts[i]*8,1,f);			
        }
		else if (th->type[i] == 1 && !th->decimal[i]) {
		    unsigned int grp_count;
		    fread(&cnt, 4, 1, f);
		    fread(th->h_columns[i],cnt*8,1,f);		
			fread(&grp_count, 4, 1, f);
		}	
        else {
            unsigned int grp_count;
            CudaChar *c = (CudaChar*)th->h_columns[i];

            fread(&cnt, 4, 1, f);
            if(!c->compressed) {
                if( hipSuccess != hipHostMalloc(&(c->compressed), cnt*8)) {
                    cout << "couldn't allocate " << cnt*8 << " chars " << endl;
                    exit(-1);
                };
            };
            fread(c->compressed,cnt*8,1,f);
            fread(&grp_count, 4, 1, f);
            fread(&grp_count, 4, 1, f);
            fread(&grp_count, 4, 1, f);
            for(int j = 0; j < c->mColumnCount; j++)
                fread(c->h_columns[j],grp_count,1,f);
        };
    };
    buffersLoaded = 1;

}




unsigned int findSegmentCount(char* file_name)
{
    unsigned int orig_recCount;
    unsigned int comp_type, cnt;


    FILE* f;
    f = fopen ( file_name , "rb" );
    if (f==NULL) {
        cout << "Cannot open file " << file_name << endl;
        exit (1);
    }
    fread(&cnt, 4, 1, f);
    fseeko(f, cnt*8 + 16, SEEK_CUR);
    fread(&comp_type, 4, 1, f);
    if(comp_type == 2)
        fread(&orig_recCount, 4, 1, f);
	else if(comp_type == 3)	
	    orig_recCount = cnt;
    else {
        fread(&orig_recCount, 4, 1, f);
        fread(&orig_recCount, 4, 1, f);
    };

    fclose(f);

    return orig_recCount;
};



long long int findRecordCount(char* file_name, unsigned int mColumnCount, unsigned int& segCount, unsigned int& maxRecs)
{
    FILE* f;
    unsigned int grp_count;
    f = fopen ( file_name , "rb" );
    if (f==NULL) {
        cout << "Cannot open file " << file_name << endl;
        exit (1);
    }


    long long int RecCount = 0;
	segCount = 0;
	maxRecs = 0;

    unsigned int bits, cnt, fit_count, orig_recCount;
    int_type orig_lower_val;
    unsigned int comp_type;
    long long int start_val;

    while (!feof(f)) {
        fread(&cnt, 4, 1, f);
        if (feof(f)) {
		    fclose(f);
		    return RecCount;
		}
	    segCount++;
        fseeko(f, cnt*8 + 16 , SEEK_CUR);
        fread(&comp_type, 4, 1, f);

        if(comp_type == 3) {
		
		}
		else {
            if(comp_type == 2) {
                fread(&orig_recCount, 4, 1, f);
                fread(&grp_count, 4, 1, f);
                fseeko(f, grp_count*mColumnCount , SEEK_CUR);
                fread(&grp_count, 4, 3, f);
            }		 
            else {
                fread(&orig_recCount, 4, 1, f);
                fread(&orig_recCount, 4, 1, f);
            };
			
       		if (orig_recCount > maxRecs) 
		            maxRecs = orig_recCount;								
			
            RecCount = RecCount + orig_recCount;
            fread(&bits, 4, 1 ,f);
            fread(&orig_lower_val, 8, 1, f);
            fread(&fit_count, 4, 1 ,f);
            fread((char *)&start_val, 8, 1, f);
            fread((char *)&comp_type, 4, 1, f);
		};	

    };
    fclose(f);
    //cout << "found reccount " << RecCount << endl;
    return RecCount;


}

